#include "hip/hip_runtime.h"

#include "customAllReduceKernels.h"
#include "../common/cudaBf16Fallbacks.cuh"
#include "../common/cudaTypeUtils.cuh"
#include "../common/cudaUtils.h"
#include "../common/customAllReduceUtils.h"
#include "../common/dataType.h"
#include "../common/envUtils.h"
#include <hip/hip_cooperative_groups.h>
#include <tuple>
#include <type_traits>

namespace sugesstify::kernels
{

using sugesstify::common::divUp;
using sugesstify::common::roundUp;


static inline __device__ void st_flag_release(uint32_t const& flag, uint32_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#else
    __threadfence_system();
    asm volatile("st.global.volatile.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#endif
}


static inline __device__ uint32_t ld_flag_acquire(uint32_t* flag_addr)
{
    uint32_t flag;
#if __CUDA_ARCH__ >= 700
    asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
    asm volatile("ld.global.volatile.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#endif
    return flag;
}


using PackedFloat = union
{
    int4 packed;
    float unpacked[4];
};

using PackedHalf = union
{
    int4 packed;
    half2 unpacked[4];
};

template <typename T>
struct PackedOn16Bytes
{
};

template <>
struct PackedOn16Bytes<float>
{
    using Type = PackedFloat;
};

template <>
struct PackedOn16Bytes<half>
{
    using Type = PackedHalf;
};

#ifdef ENABLE_BF16
using PackedBFloat16 = union
{
    int4 packed;
    __hip_bfloat162 unpacked[4];
};

template <>
struct PackedOn16Bytes<__hip_bfloat16>
{
    using Type = PackedBFloat16;
};

#endif

template <typename T>
inline __device__ int4 add128b(T& a, T& b)
{
    T c;
    c.unpacked[0] = a.unpacked[0] + b.unpacked[0];
    c.unpacked[1] = a.unpacked[1] + b.unpacked[1];
    c.unpacked[2] = a.unpacked[2] + b.unpacked[2];
    c.unpacked[3] = a.unpacked[3] + b.unpacked[3];
    return c.packed;
}

__inline__ __device__ void multi_gpu_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
    size_t const world_size, int const tidx, int const bidx)
{
    if (tidx < world_size)
    {

        size_t offset = (flag % 2) ? world_size : 0;

        if (bidx == 0)
        {
            st_flag_release(flag, signals[tidx] + offset + local_rank);
        }

        uint32_t* peer_barrier_d = signals[local_rank] + offset + tidx;
        while (ld_flag_acquire(peer_barrier_d) != flag)
        {
        }
    }

    __syncthreads();
}

__inline__ __device__ void block_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
    size_t const world_size, int const tidx, int const bidx, int const grid_size)
{
    if (tidx < world_size)
    {

        uint32_t flag_block_offset = world_size + bidx * world_size;

        if (flag % 2 == 1)
        {
            flag_block_offset += (grid_size + 1) * world_size;
        }

        st_flag_release(flag, signals[tidx] + flag_block_offset + local_rank);

        uint32_t* peer_barrier_d = signals[local_rank] + flag_block_offset + tidx;

        while (ld_flag_acquire(peer_barrier_d) != flag)
        {
        }
    }

    __syncthreads();
}

namespace reduce_fusion
{

inline __device__ float warp_reduce_sum(float val)
{
    val += __shfl_xor_sync(~0, val, 16);
    val += __shfl_xor_sync(~0, val, 8);
    val += __shfl_xor_sync(~0, val, 4);
    val += __shfl_xor_sync(~0, val, 2);
    val += __shfl_xor_sync(~0, val, 1);
    return val;
}

inline __device__ float block_reduce_sum(float val)
{
    __shared__ float smem[details::kWarpSize];
    int lane_id = threadIdx.x % details::kWarpSize, warp_id = threadIdx.x / details::kWarpSize,
        warp_num = blockDim.x / details::kWarpSize;
    val = warp_reduce_sum(val);
    if (lane_id == 0)
    {
        smem[warp_id] = val;
    }
    __syncthreads();
    val = lane_id < warp_num ? smem[lane_id] : 0.f;
    val = warp_reduce_sum(val);
    return val;
}

template <typename T, typename PackedStruct>
inline __device__ float accumulate(float acc, PackedStruct& vec)
{
    static constexpr int kLoopNum = sizeof(PackedStruct) / sizeof(T);
#pragma unroll
    for (int i = 0; i < kLoopNum; ++i)
    {
        float v = static_cast<float>(reinterpret_cast<T*>(vec.unpacked)[i]);
        acc += v * v;
    }
    return acc;
}

template <typename T, bool Affine, typename PackedStruct>
inline __device__ int4 rms_norm(float denom, PackedStruct& vec, PackedStruct& weight)
{
    static constexpr int kLoopNum = sizeof(PackedStruct) / sizeof(T);
    PackedStruct ret;
#pragma unroll
    for (int i = 0; i < kLoopNum; ++i)
    {
        float v1 = static_cast<float>(reinterpret_cast<T*>(vec.unpacked)[i]);
        if constexpr (Affine)
        {
            float v2 = static_cast<float>(reinterpret_cast<T*>(weight.unpacked)[i]);
            reinterpret_cast<T*>(ret.unpacked)[i] = static_cast<T>(__fdividef(v1, denom) * v2);
        }
        else
        {
            reinterpret_cast<T*>(ret.unpacked)[i] = static_cast<T>(__fdividef(v1, denom));
        }
    }
    return ret.packed;
}

template <typename T, bool Bias = false, bool Residual = false, bool Affine = false, bool UseSmem = false>
__global__ void rms_norm_kernel(AllReduceParams params)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    extern __shared__ uint8_t smem_ptr[];
    T* smem = reinterpret_cast<T*>(smem_ptr);

    int bid = blockIdx.x, tid = threadIdx.x;

    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    if constexpr (Residual)
    {
        residual_buffer += block_offset;
    }
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    PackedStruct inter_vec, weight_vec;
    float acc = 0.f;
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        inter_vec.packed = *reinterpret_cast<int4 const*>(intermediate_buffer + offset);
        if constexpr (Bias)
        {
            PackedStruct bias_vec;
            bias_vec.packed = *reinterpret_cast<int4 const*>(bias_buffer + offset);
            inter_vec.packed = add128b(inter_vec, bias_vec);
        }
        if constexpr (Residual)
        {
            PackedStruct residual_vec;
            residual_vec.packed = *reinterpret_cast<int4 const*>(residual_buffer + offset);
            inter_vec.packed = add128b(inter_vec, residual_vec);
            *reinterpret_cast<int4*>(intermediate_buffer + offset) = inter_vec.packed;
        }
        acc = accumulate<T>(acc, inter_vec);
        if constexpr (UseSmem)
        {
            *reinterpret_cast<int4*>(&smem[offset]) = inter_vec.packed;
        }
    }
    acc = block_reduce_sum(acc);
    float denom = __fsqrt_rn(__fdividef(acc, params.fusion_params.hidden_size) + params.fusion_params.eps);
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        if constexpr (UseSmem)
        {
            inter_vec.packed = *reinterpret_cast<int4 const*>(&smem[offset]);
        }
        if constexpr (Affine)
        {
            weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + offset);
        }
        inter_vec.packed = rms_norm<T, Affine>(denom, inter_vec, weight_vec);
        *reinterpret_cast<int4*>(&local_final_output_buffer[offset]) = inter_vec.packed;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, bool Bias = false, bool Residual = false, bool Affine = false>
__global__ void rms_pre_post_norm_kernel(AllReduceParams params)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    int bid = blockIdx.x, tid = threadIdx.x;

    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T const* weight_buffer_pre_residual_norm
        = reinterpret_cast<T const*>(params.fusion_params.weight_buffer_pre_residual_norm);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    if constexpr (Residual)
    {
        residual_buffer += block_offset;
    }
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    PackedStruct inter_vec, weight_vec, weight_vec_pre_residual_norm, bias_vec;
    float acc = 0.f;
    float acc_pre_residual_norm = 0.f;
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        inter_vec.packed = *reinterpret_cast<int4 const*>(intermediate_buffer + offset);
        if constexpr (Bias)
        {
            bias_vec.packed = *reinterpret_cast<int4 const*>(bias_buffer + offset);
        }

        if constexpr (Bias)
        {
            inter_vec.packed = add128b(inter_vec, bias_vec);
        }

        acc_pre_residual_norm = accumulate<T>(acc_pre_residual_norm, inter_vec);
        acc_pre_residual_norm = block_reduce_sum(acc_pre_residual_norm);
        float denom_pre_residual_norm = __fsqrt_rn(
            __fdividef(acc_pre_residual_norm, params.fusion_params.hidden_size) + params.fusion_params.eps);

        if constexpr (Affine)
        {
            weight_vec_pre_residual_norm.packed
                = *reinterpret_cast<int4 const*>(weight_buffer_pre_residual_norm + thread_offset);
        }
        inter_vec.packed = rms_norm<T, Affine>(denom_pre_residual_norm, inter_vec, weight_vec_pre_residual_norm);

        if constexpr (Residual)
        {
            PackedStruct residual_vec;
            residual_vec.packed = *reinterpret_cast<int4 const*>(residual_buffer + offset);
            inter_vec.packed = add128b(inter_vec, residual_vec);
            *reinterpret_cast<int4*>(intermediate_buffer + offset) = inter_vec.packed;
        }
        acc = accumulate<T>(acc, inter_vec);
    }
    acc = block_reduce_sum(acc);
    float denom = __fsqrt_rn(__fdividef(acc, params.fusion_params.hidden_size) + params.fusion_params.eps);
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        if constexpr (Affine)
        {
            weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + offset);
        }
        inter_vec.packed = rms_norm<T, Affine>(denom, inter_vec, weight_vec);
        *reinterpret_cast<int4*>(&local_final_output_buffer[offset]) = inter_vec.packed;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, bool Bias = false, bool Residual = false, bool Affine = false>
void rms_norm_kernel_launcher(AllReduceParams& params, hipStream_t stream, AllReduceFusionOp fusionOp)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    CHECK(params.fusion_params.hidden_size % kPackedSize == 0);
    if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM)
    {
        CHECK(params.fusion_params.hidden_size <= 8192);
    }
    int need_threads = params.fusion_params.hidden_size / kPackedSize;
    int cta_size;
    if (need_threads <= details::kMaxCtaSize)
    {
        cta_size = (need_threads + details::kWarpSize - 1) / details::kWarpSize * details::kWarpSize;
    }
    else
    {
        cta_size = details::kMaxCtaSize;
    }
    int cta_num = params.elts_total / params.fusion_params.hidden_size;
    int smem_size = 0;
    if (cta_size * details::kBytesPerAccess / sizeof(T) < params.fusion_params.hidden_size)
    {
        smem_size = params.fusion_params.hidden_size * sizeof(T);
        if (sugesstify::common::getEnvEnablePDL())
        {
            LOG_DEBUG("Enable PDL in rms_norm_kernel");
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = cta_num;
            kernelConfig.blockDim = cta_size;
            kernelConfig.dynamicSmemBytes = smem_size;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
            {
                CUDA_CHECK(
                    cudaLaunchKernelEx(&kernelConfig, rms_norm_kernel<T, Bias, Residual, Affine, true>, params));
            }
            else
            {
                CUDA_CHECK(
                    cudaLaunchKernelEx(&kernelConfig, rms_pre_post_norm_kernel<T, Bias, Residual, Affine>, params));
            }
        }
        else
        {
            if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
            {
                rms_norm_kernel<T, Bias, Residual, Affine, true><<<cta_num, cta_size, smem_size, stream>>>(params);
            }
            else
            {
                rms_pre_post_norm_kernel<T, Bias, Residual, Affine><<<cta_num, cta_size, smem_size, stream>>>(params);
            }
        }
    }
    else
    {
        if (sugesstify::common::getEnvEnablePDL())
        {
            LOG_DEBUG("Enable PDL in rms_norm_kernel");
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = cta_num;
            kernelConfig.blockDim = cta_size;
            kernelConfig.dynamicSmemBytes = smem_size;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
            {
                CUDA_CHECK(
                    cudaLaunchKernelEx(&kernelConfig, rms_norm_kernel<T, Bias, Residual, Affine, false>, params));
            }
            else
            {
                CUDA_CHECK(
                    cudaLaunchKernelEx(&kernelConfig, rms_pre_post_norm_kernel<T, Bias, Residual, Affine>, params));
            }
        }
        else
        {
            if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
            {
                rms_norm_kernel<T, Bias, Residual, Affine, false><<<cta_num, cta_size, smem_size, stream>>>(params);
            }
            else
            {
                rms_pre_post_norm_kernel<T, Bias, Residual, Affine><<<cta_num, cta_size, smem_size, stream>>>(params);
            }
        }
    }
}

template <typename T>
struct NegZero128b
{
    static constexpr int v = static_cast<int>(0x80008000);
    static constexpr int4 value = {v, v, v, v};
};

template <>
struct NegZero128b<float>
{
    static constexpr int v = static_cast<int>(0x80000000);
    static constexpr int4 value = {v, v, v, v};
};

template <typename T>
__device__ static constexpr int4 NegZero128b_v = NegZero128b<T>::value;

template <typename T>
__device__ __forceinline__ bool is_neg_zero(T& v);

template <>
__device__ __forceinline__ bool is_neg_zero<float>(float& v)
{
    uint32_t bits = *reinterpret_cast<uint32_t*>(&v);
    return bits == 0x80000000;
}

template <>
__device__ __forceinline__ bool is_neg_zero<half>(half& v)
{
    uint16_t bits = *reinterpret_cast<uint16_t*>(&v);
    return bits == 0x8000;
}

template <>
__device__ __forceinline__ bool is_neg_zero<__hip_bfloat16>(__hip_bfloat16& v)
{
    uint16_t bits = *reinterpret_cast<uint16_t*>(&v);
    return bits == 0x8000;
}

template <typename ValType, typename VecType>
__device__ __forceinline__ VecType remove_neg_zero(VecType const& vec)
{
    static constexpr int kIter = sizeof(VecType) / sizeof(ValType);
    using ReadOnlyValType = std::add_const_t<ValType>;
    VecType ret;
#pragma unroll
    for (int i = 0; i < kIter; ++i)
    {
        auto val = reinterpret_cast<ReadOnlyValType*>(&vec)[i];
        reinterpret_cast<ValType*>(&ret)[i] = is_neg_zero(val) ? static_cast<ValType>(0.f) : val;
    }
    return ret;
}

template <typename ValType, typename VecType>
__device__ __forceinline__ bool has_neg_zero(VecType const& vec)
{
    static constexpr int kIter = sizeof(VecType) / sizeof(ValType);
    using ReadOnlyValType = std::add_const_t<ValType>;
#pragma unroll
    for (int i = 0; i < kIter; ++i)
    {
        auto val = reinterpret_cast<ReadOnlyValType*>(&vec)[i];
        if (is_neg_zero(val))
        {
            return true;
        }
    }
    return false;
}

template <typename ValType, typename VecType>
__device__ __forceinline__ bool all_neg_zero(VecType const& vec)
{
    static constexpr int kIter = sizeof(VecType) / sizeof(ValType);
    using ReadOnlyValType = std::add_const_t<ValType>;
#pragma unroll
    for (int i = 0; i < kIter; ++i)
    {
        auto val = reinterpret_cast<ReadOnlyValType*>(&vec)[i];
        if (!is_neg_zero(val))
        {
            return false;
        }
    }
    return true;
}

__device__ __forceinline__ void st_global_release(int4 const& val, int4* addr)
{
    asm volatile("st.release.global.sys.v4.b32 [%4], {%0, %1, %2, %3};" ::"r"(val.x), "r"(val.y), "r"(val.z),
        "r"(val.w), "l"(addr));
}

__device__ __forceinline__ int4 ld_global_acquire(int4* addr)
{
    int4 val;
    asm volatile("ld.acquire.global.sys.v4.b32 {%0, %1, %2, %3}, [%4];"
                 : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                 : "l"(addr));
    return val;
}

__device__ __forceinline__ void st_global_volatile(int4 const& val, int4* addr)
{
    asm volatile("st.volatile.global.v4.b32 [%4], {%0, %1, %2, %3};" ::"r"(val.x), "r"(val.y), "r"(val.z), "r"(val.w),
        "l"(addr));
}

__device__ __forceinline__ int4 ld_global_volatile(int4* addr)
{
    int4 val;
    asm volatile("ld.volatile.global.v4.b32 {%0, %1, %2, %3}, [%4];"
                 : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                 : "l"(addr));
    return val;
}

template <typename ValType>
__device__ __forceinline__ void set_neg_zero(int4* addr)
{
    st_global_volatile(NegZero128b_v<ValType>, addr);
}

template <typename T, int RanksPerNode, bool PushMode>
struct Reducer;

template <typename T, int RanksPerNode>
struct Reducer<T, RanksPerNode, true>
{
    static __device__ __forceinline__ int4 allreduce(AllReduceParams& params, int global_offset)
    {
        using PackedStruct = typename PackedOn16Bytes<T>::Type;
        int ping = params.barrier_flag % 3;
        int pong = (params.barrier_flag + 2) % 3;
        T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
        T* local_shared_buffer = reinterpret_cast<T*>(
            params.fusion_params.lamport_peer_comm_buffer_ptrs[params.local_rank + ping * MAX_RANKS_PER_NODE]);
        T* local_clean_buffer = reinterpret_cast<T*>(
            params.fusion_params.lamport_peer_comm_buffer_ptrs[params.local_rank + pong * MAX_RANKS_PER_NODE]);
        local_input_buffer += global_offset;
        local_shared_buffer += global_offset;
        local_clean_buffer += global_offset;
        T* buffers[RanksPerNode];
#pragma unroll
        for (int ii = 0; ii < RanksPerNode; ++ii)
        {
            int rank = (params.local_rank + ii) % RanksPerNode;
            buffers[ii] = reinterpret_cast<T*>(
                              params.fusion_params.lamport_peer_comm_buffer_ptrs[rank + ping * MAX_RANKS_PER_NODE])
                + global_offset + params.local_rank * params.elts_total;
        }
        PackedStruct sum_vec, val;
        val.packed = remove_neg_zero<T>(*reinterpret_cast<int4 const*>(local_input_buffer));
#pragma unroll
        for (int ii = 1; ii < RanksPerNode; ++ii)
        {
            st_global_volatile(val.packed, reinterpret_cast<int4*>(buffers[ii]));
        }
        sum_vec.packed = val.packed;
#pragma unroll
        for (int ii = 1; ii < RanksPerNode; ++ii)
        {
            int rank = (params.local_rank + ii) % RanksPerNode;
            set_neg_zero<T>(reinterpret_cast<int4*>(local_clean_buffer + rank * params.elts_total));
        }
        PackedStruct vals[RanksPerNode - 1];
        bool done = false;
        while (!done)
        {
            done = true;
#pragma unroll
            for (int ii = 1; ii < RanksPerNode; ++ii)
            {
                int rank = (params.local_rank + ii) % RanksPerNode;
                vals[ii - 1].packed
                    = ld_global_volatile(reinterpret_cast<int4*>(local_shared_buffer + rank * params.elts_total));
            }
#pragma unroll
            for (int ii = 0; ii < RanksPerNode - 1; ii++)
            {
                done &= !has_neg_zero<T>(vals[ii].packed);
            }
        }

#pragma unroll
        for (int ii = 1; ii < RanksPerNode; ++ii)
        {
            sum_vec.packed = add128b(sum_vec, vals[ii - 1]);
        }
        return sum_vec.packed;
    }
};

template <typename T, int RanksPerNode>
struct Reducer<T, RanksPerNode, false>
{
    static __device__ __forceinline__ int4 allreduce(AllReduceParams& params, int global_offset)
    {
        using PackedStruct = typename PackedOn16Bytes<T>::Type;
        int ping = params.barrier_flag % 3;
        int pong = (params.barrier_flag + 2) % 3;
        T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
        T* local_shared_buffer = reinterpret_cast<T*>(
            params.fusion_params.lamport_peer_comm_buffer_ptrs[params.local_rank + ping * MAX_RANKS_PER_NODE]);
        T* local_clean_buffer = reinterpret_cast<T*>(
            params.fusion_params.lamport_peer_comm_buffer_ptrs[params.local_rank + pong * MAX_RANKS_PER_NODE]);
        local_input_buffer += global_offset;
        local_shared_buffer += global_offset;
        local_clean_buffer += global_offset;
        T* buffers[RanksPerNode];
#pragma unroll
        for (int ii = 0; ii < RanksPerNode; ++ii)
        {
            int rank = (params.local_rank + ii) % RanksPerNode;
            buffers[ii] = reinterpret_cast<T*>(
                              params.fusion_params.lamport_peer_comm_buffer_ptrs[rank + ping * MAX_RANKS_PER_NODE])
                + global_offset;
        }
        PackedStruct sum_vec, val;
        val.packed = remove_neg_zero<T>(*reinterpret_cast<int4 const*>(local_input_buffer));
        st_global_volatile(val.packed, reinterpret_cast<int4*>(local_shared_buffer));
        sum_vec.packed = val.packed;
#pragma unroll
        for (int ii = 1; ii < RanksPerNode; ++ii)
        {
            do
            {
                val.packed = ld_global_volatile(reinterpret_cast<int4*>(buffers[ii]));
            } while (has_neg_zero<T>(val.packed));
            sum_vec.packed = add128b(sum_vec, val);
        }
        set_neg_zero<T>(reinterpret_cast<int4*>(local_clean_buffer));
        return sum_vec.packed;
    }
};

template <int ClusterSize, typename T, int RanksPerNode, bool Bias = false, bool Affine = false, bool PushMode = true>
static __global__ void lamport_style_one_shot_all_reduce_norm_kernel(AllReduceParams params)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    namespace cg = cooperative_groups;
    static_assert(RanksPerNode <= 8);
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    cg::cluster_group cluster = cg::this_cluster();

    __shared__ float cluster_acc, cluster_acc_sum;

    int bid = blockIdx.x, tid = threadIdx.x;
    int cluster_id = bid / ClusterSize, cluster_block_rank = bid % ClusterSize;

    int token_id = cluster_id;
    int cluster_offset = token_id * params.fusion_params.hidden_size;
    int block_offset = cluster_block_rank * params.fusion_params.hidden_size / ClusterSize;
    int thread_offset = tid * kPackedSize;

    int inner_token_offset = block_offset + thread_offset;
    int global_offset = cluster_offset + inner_token_offset;

    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    local_final_output_buffer += global_offset;
    intermediate_buffer += global_offset;
    residual_buffer += global_offset;
    bias_buffer += inner_token_offset;
    weight_buffer += inner_token_offset;

    PackedStruct weight_vec, bias_vec, residual_vec;
    residual_vec.packed = *reinterpret_cast<int4 const*>(residual_buffer);
    if constexpr (Bias)
    {
        bias_vec.packed = *reinterpret_cast<int4 const*>(bias_buffer);
    }
    if constexpr (Affine)
    {
        weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer);
    }

    cudaGridDependencySynchronize();

    float acc = 0.f;
    PackedStruct sum_vec;
    sum_vec.packed = Reducer<T, RanksPerNode, PushMode>::allreduce(params, global_offset);

    if constexpr (Bias)
    {
        sum_vec.packed = add128b(sum_vec, bias_vec);
    }
    sum_vec.packed = add128b(sum_vec, residual_vec);
    *reinterpret_cast<int4*>(intermediate_buffer) = sum_vec.packed;
    acc = accumulate<T>(acc, sum_vec);
    acc = block_reduce_sum(acc);
    if (ClusterSize > 1)
    {
        if (threadIdx.x == 0)
        {
            cluster_acc = acc;
        }
        cluster.sync();
        if (threadIdx.x == 0)
        {
            acc = 0.f;
#pragma unroll
            for (int ii = 0; ii < ClusterSize; ++ii)
            {
                acc += *cluster.map_shared_rank(&cluster_acc, ii);
            }
            cluster_acc_sum = acc;
        }
        __syncthreads();
        acc = cluster_acc_sum;
        cluster.sync();
    }

    float denom = __fsqrt_rn(__fdividef(acc, params.fusion_params.hidden_size) + params.fusion_params.eps);
    sum_vec.packed = rms_norm<T, Affine>(denom, sum_vec, weight_vec);
    *reinterpret_cast<int4*>(local_final_output_buffer) = sum_vec.packed;

    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

int heuristic_min_warp_number(int tp_size, int hidden_size)
{
    if (hidden_size >= 4096)
    {
        return 4;
    }
    if (tp_size == 2)
    {
        return 32;
    }
    else
    {
        return 16;
    }
}

template <typename T, int RanksPerNode, bool Bias, bool Affine>
void lamport_style_one_shot_all_reduce_norm_kernel_launcher(AllReduceParams params, hipStream_t stream)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    CHECK(params.fusion_params.hidden_size % kPackedSize == 0);
    int threads_per_token = params.fusion_params.hidden_size / kPackedSize;
    int warps_per_token = (threads_per_token + details::kWarpSize - 1) / details::kWarpSize;
    int token_num = params.elts_total / params.fusion_params.hidden_size;
    int warp_min_number = heuristic_min_warp_number(RanksPerNode, params.fusion_params.hidden_size);
    int cluster_size = std::min(((warps_per_token + warp_min_number - 1) / warp_min_number), details::kClusterMaxSize);
    int cta_size = warps_per_token / cluster_size * details::kWarpSize;
    CHECK(cta_size <= details::kMaxCtaSize);
    int cta_num = token_num * cluster_size;
    hipLaunchConfig_t kernel_config = {0};
    kernel_config.gridDim = cta_num;
    kernel_config.blockDim = cta_size;
    kernel_config.dynamicSmemBytes = 0;
    kernel_config.stream = stream;

    hipLaunchAttribute attribute[2];
    attribute[0].id = cudaLaunchAttributeClusterDimension;
    attribute[0].val.clusterDim.x = cluster_size;
    attribute[0].val.clusterDim.y = 1;
    attribute[0].val.clusterDim.z = 1;
    kernel_config.attrs = attribute;
    kernel_config.numAttrs = 1;
    if (sugesstify::common::getEnvEnablePDL())
    {
        attribute[1].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attribute[1].val.programmaticStreamSerializationAllowed = 1;
        kernel_config.numAttrs++;
    }
#define LAUNCH_LAMPORT_KERNEL(CLUSTER_SIZE)                                                                            \
    if (cluster_size == CLUSTER_SIZE)                                                                                  \
    {                                                                                                                  \
        CUDA_CHECK(cudaLaunchKernelEx(&kernel_config,                                                             \
            lamport_style_one_shot_all_reduce_norm_kernel<CLUSTER_SIZE, T, RanksPerNode, Bias, Affine>, params));      \
        return;                                                                                                        \
    }
    LAUNCH_LAMPORT_KERNEL(1);
    LAUNCH_LAMPORT_KERNEL(2);
    LAUNCH_LAMPORT_KERNEL(3);
    LAUNCH_LAMPORT_KERNEL(4);
    LAUNCH_LAMPORT_KERNEL(5);
    LAUNCH_LAMPORT_KERNEL(6);
    LAUNCH_LAMPORT_KERNEL(7);
    LAUNCH_LAMPORT_KERNEL(8);
#undef LAUNCH_LAMPORT_KERNEL
}

template <typename T, int RanksPerNode, bool Bias = false, bool Affine = false, bool UseSmem = false>
static __global__ void __launch_bounds__(1024, 1) one_shot_all_reduce_norm_kernel(AllReduceParams params)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    extern __shared__ uint8_t smem_ptr[];
    T* smem = reinterpret_cast<T*>(smem_ptr);

    int bid = blockIdx.x, tid = threadIdx.x;
    int norm_num = params.elts_total / params.fusion_params.hidden_size;
    int norm_per_block = (norm_num + gridDim.x - 1) / gridDim.x;
    int norm_this_block = std::min(norm_per_block, norm_num - bid * norm_per_block);

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * norm_per_block * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    local_input_buffer += block_offset;
    residual_buffer += block_offset;
    local_shared_buffer += block_offset;
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

    T* buffers[RanksPerNode];
#pragma unroll
    for (int ii = 0; ii < RanksPerNode; ++ii)
    {
        int rank = (params.local_rank + ii) % RanksPerNode;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    for (int offset = thread_offset; offset < norm_this_block * params.fusion_params.hidden_size;
         offset += blockDim.x * kPackedSize)
    {
        *reinterpret_cast<int4*>(&local_shared_buffer[offset])
            = *reinterpret_cast<int4 const*>(&local_input_buffer[offset]);
    }
    block_barrier(
        params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RanksPerNode, tid, bid, gridDim.x);
    for (int norm_idx = 0; norm_idx < norm_this_block; ++norm_idx)
    {
        int norm_offset = norm_idx * params.fusion_params.hidden_size;
        float acc = 0.f;
        PackedStruct sum_vec, weight_vec, bias_vec, residual_vec;
        for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
        {
            PackedStruct vals[RanksPerNode];
            sum_vec.packed = {0, 0, 0, 0};
            if constexpr (Bias)
            {
                bias_vec.packed = *reinterpret_cast<int4 const*>(&bias_buffer[offset]);
            }
            residual_vec.packed = *reinterpret_cast<int4 const*>(&residual_buffer[norm_offset + offset]);
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][block_offset + norm_offset + offset]);
            }
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                sum_vec.packed = add128b(sum_vec, vals[ii]);
            }
            if constexpr (Bias)
            {
                sum_vec.packed = add128b(sum_vec, bias_vec);
            }
            sum_vec.packed = add128b(sum_vec, residual_vec);
            *reinterpret_cast<int4*>(&intermediate_buffer[norm_offset + offset]) = sum_vec.packed;
            acc = accumulate<T>(acc, sum_vec);
            if constexpr (UseSmem)
            {
                *reinterpret_cast<int4*>(&smem[offset]) = sum_vec.packed;
            }
        }
        acc = block_reduce_sum(acc);
        float denom = __fsqrt_rn(__fdividef(acc, params.fusion_params.hidden_size) + params.fusion_params.eps);
        for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
        {
            if constexpr (UseSmem)
            {
                sum_vec.packed = *reinterpret_cast<int4 const*>(&smem[offset]);
            }
            if constexpr (Affine)
            {
                weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + offset);
            }
            sum_vec.packed = rms_norm<T, Affine>(denom, sum_vec, weight_vec);
            *reinterpret_cast<int4*>(&local_final_output_buffer[norm_offset + offset]) = sum_vec.packed;
        }
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, int RanksPerNode, bool Bias = false, bool Affine = false>
static __global__ void __launch_bounds__(1024, 1) one_shot_prenorm_all_reduce_norm_kernel(AllReduceParams params)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    int bid = blockIdx.x, tid = threadIdx.x;
    int norm_num = params.elts_total / params.fusion_params.hidden_size;
    int norm_per_block = (norm_num + gridDim.x - 1) / gridDim.x;
    int norm_this_block = std::min(norm_per_block, norm_num - bid * norm_per_block);

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T const* weight_buffer_pre_residual_norm
        = reinterpret_cast<T const*>(params.fusion_params.weight_buffer_pre_residual_norm);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * norm_per_block * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    local_input_buffer += block_offset;
    residual_buffer += block_offset;
    local_shared_buffer += block_offset;
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

    T* buffers[RanksPerNode];
#pragma unroll
    for (int ii = 0; ii < RanksPerNode; ++ii)
    {
        int rank = (params.local_rank + ii) % RanksPerNode;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    for (int offset = thread_offset; offset < norm_this_block * params.fusion_params.hidden_size;
         offset += blockDim.x * kPackedSize)
    {
        *reinterpret_cast<int4*>(&local_shared_buffer[offset])
            = *reinterpret_cast<int4 const*>(&local_input_buffer[offset]);
    }
    block_barrier(
        params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RanksPerNode, tid, bid, gridDim.x);
    for (int norm_idx = 0; norm_idx < norm_this_block; ++norm_idx)
    {
        int norm_offset = norm_idx * params.fusion_params.hidden_size;
        float acc = 0.f;
        float acc_pre_residual_norm = 0.f;
        PackedStruct sum_vec, weight_vec, bias_vec, residual_vec, weight_vec_pre_residual_norm;
        for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
        {
            PackedStruct vals[RanksPerNode];
            sum_vec.packed = {0, 0, 0, 0};
            if constexpr (Bias)
            {
                bias_vec.packed = *reinterpret_cast<int4 const*>(&bias_buffer[offset]);
            }
            residual_vec.packed = *reinterpret_cast<int4 const*>(&residual_buffer[norm_offset + offset]);
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][block_offset + norm_offset + offset]);
            }
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                sum_vec.packed = add128b(sum_vec, vals[ii]);
            }

            if constexpr (Bias)
            {
                sum_vec.packed = add128b(sum_vec, bias_vec);
            }

            acc_pre_residual_norm = accumulate<T>(acc_pre_residual_norm, sum_vec);

            acc_pre_residual_norm = block_reduce_sum(acc_pre_residual_norm);

            float denom_pre_residual_norm = __fsqrt_rn(
                __fdividef(acc_pre_residual_norm, params.fusion_params.hidden_size) + params.fusion_params.eps);
            if constexpr (Affine)
            {
                weight_vec_pre_residual_norm.packed
                    = *reinterpret_cast<int4 const*>(weight_buffer_pre_residual_norm + thread_offset);
            }
            sum_vec.packed = rms_norm<T, Affine>(denom_pre_residual_norm, sum_vec, weight_vec_pre_residual_norm);

            sum_vec.packed = add128b(sum_vec, residual_vec);
            *reinterpret_cast<int4*>(&intermediate_buffer[norm_offset + offset]) = sum_vec.packed;
            acc = accumulate<T>(acc, sum_vec);
        }
        acc = block_reduce_sum(acc);
        float denom = __fsqrt_rn(__fdividef(acc, params.fusion_params.hidden_size) + params.fusion_params.eps);
        if constexpr (Affine)
        {
            weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + thread_offset);
        }
        sum_vec.packed = rms_norm<T, Affine>(denom, sum_vec, weight_vec);
        *reinterpret_cast<int4*>(&local_final_output_buffer[norm_offset + thread_offset]) = sum_vec.packed;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T>
bool is_lamport_supported(int token_num, int hidden_size)
{
    static char* disableLamportReduceNormFusionChar = std::getenv("DISABLE_LAMPORT_REDUCE_NORM_FUSION");
    bool disableLamportReduceNormFusion = (disableLamportReduceNormFusionChar != nullptr);
    if (disableLamportReduceNormFusion)
        return false;
    static int sm = sugesstify::common::getSMVersion();
    if (sm < 90)
    {
        return false;
    }
    if (!std::is_same_v<T, half> && !std::is_same_v<T, __hip_bfloat16>)
    {
        return false;
    }
    if (token_num > details::kLamportTokenNumThreshold)
    {
        return false;
    }
    if (hidden_size < details::kLamportHiddenSizeThreshold)
    {
        return false;
    }
    return true;
}

bool is_lamport_supported(nvinfer1::DataType dataType, int token_num, int hidden_size)
{
    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT: return is_lamport_supported<float>(token_num, hidden_size);
    case nvinfer1::DataType::kHALF: return is_lamport_supported<half>(token_num, hidden_size);
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: return is_lamport_supported<__hip_bfloat16>(token_num, hidden_size);
#endif
    default: return false;
    }
}

template <typename T, int RanksPerNode, bool Bias, bool Affine>
void one_shot_all_reduce_norm_kernel_launcher(AllReduceParams& params, hipStream_t stream, AllReduceFusionOp fusionOp)
{
    int token_num = params.elts_total / params.fusion_params.hidden_size;

    if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM)
    {
        CHECK(params.fusion_params.hidden_size <= 8192);
    }

    if (is_lamport_supported<T>(token_num, params.fusion_params.hidden_size)
        && (fusionOp != AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM))
    {
        lamport_style_one_shot_all_reduce_norm_kernel_launcher<T, RanksPerNode, Bias, Affine>(params, stream);
    }
    else
    {
        static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
        CHECK(params.fusion_params.hidden_size % kPackedSize == 0);
        int need_threads = params.fusion_params.hidden_size / kPackedSize;
        int cta_size;
        if (need_threads <= details::kMaxCtaSize)
        {
            cta_size = (need_threads + details::kWarpSize - 1) / details::kWarpSize * details::kWarpSize;
        }
        else
        {
            cta_size = details::kMaxCtaSize;
        }
        int norm_num = params.elts_total / params.fusion_params.hidden_size;
        int cta_num = std::min(norm_num, static_cast<int>(MAX_ALL_REDUCE_BLOCKS));
        int smem_size = 0;

        if (cta_size * kPackedSize < params.fusion_params.hidden_size)
        {
            smem_size = params.fusion_params.hidden_size * sizeof(T);
            if (sugesstify::common::getEnvEnablePDL())
            {
                LOG_DEBUG("Enable PDL in one_shot_all_reduce_norm_kernel");

                hipLaunchConfig_t kernelConfig = {0};
                kernelConfig.gridDim = cta_num;
                kernelConfig.blockDim = cta_size;
                kernelConfig.dynamicSmemBytes = smem_size;
                kernelConfig.stream = stream;

                hipLaunchAttribute attribute[1];
                attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
                attribute[0].val.programmaticStreamSerializationAllowed = 1;
                kernelConfig.attrs = attribute;
                kernelConfig.numAttrs = 1;
                if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
                {
                    CUDA_CHECK(cudaLaunchKernelEx(
                        &kernelConfig, one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, true>, params));
                }
                else
                {
                    CUDA_CHECK(cudaLaunchKernelEx(
                        &kernelConfig, one_shot_prenorm_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine>, params));
                }
            }
            else
            {
                if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
                {
                    one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, true>
                        <<<cta_num, cta_size, smem_size, stream>>>(params);
                }
                else
                {
                    one_shot_prenorm_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine>
                        <<<cta_num, cta_size, smem_size, stream>>>(params);
                }
            }
        }
        else
        {
            if (sugesstify::common::getEnvEnablePDL())
            {
                hipLaunchConfig_t kernelConfig = {0};
                kernelConfig.gridDim = cta_num;
                kernelConfig.blockDim = cta_size;
                kernelConfig.dynamicSmemBytes = smem_size;
                kernelConfig.stream = stream;

                hipLaunchAttribute attribute[1];
                attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
                attribute[0].val.programmaticStreamSerializationAllowed = 1;
                kernelConfig.attrs = attribute;
                kernelConfig.numAttrs = 1;

                LOG_DEBUG("Enable PDL in one_shot_all_reduce_norm_kernel");
                if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
                {
                    CUDA_CHECK(cudaLaunchKernelEx(
                        &kernelConfig, one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, false>, params));
                }
                else
                {
                    CUDA_CHECK(cudaLaunchKernelEx(
                        &kernelConfig, one_shot_prenorm_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine>, params));
                }
            }
            else
            {
                if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
                {
                    one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, false>
                        <<<cta_num, cta_size, smem_size, stream>>>(params);
                }
                else
                {
                    one_shot_prenorm_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine>
                        <<<cta_num, cta_size, smem_size, stream>>>(params);
                }
            }
        }
    }
}

template <typename T>
__global__ void lamport_initialize_kernel(T* buffer, size_t size)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;
    for (size_t offset = (blockIdx.x * blockDim.x + threadIdx.x) * kPackedSize; offset < size;
         offset += gridDim.x * blockDim.x * kPackedSize)
    {
        set_neg_zero<T>(reinterpret_cast<int4*>(&buffer[offset]));
    }
}

template <typename T>
void lamport_initialize_kernel_launcher(void* buffer, size_t size, hipStream_t stream)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    int block_size = 1024;
    int grid_size = (size + 1024 * kPackedSize - 1) / (1024 * kPackedSize);
    lamport_initialize_kernel<T><<<grid_size, block_size, 0, stream>>>(reinterpret_cast<T*>(buffer), size);
}
};

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true, bool PUSH_MODE = false>
static __global__ void oneShotAllReduceKernel(AllReduceParams params)
{

    int const bidx = blockIdx.x;
    int const tidx = threadIdx.x;
    int const grid_size = gridDim.x;

    static constexpr int PACKED_ELTS = 16 / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);

    size_t const chunk_start = bidx * params.elts_per_block + tidx * PACKED_ELTS;
    size_t const chunk_end = std::min((bidx + 1) * params.elts_per_block, params.elts_total);

    T* buffers[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

    if constexpr (PUSH_MODE || COPY_INPUT)
    {
        for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * PACKED_ELTS)
        {
            if constexpr (PUSH_MODE)
            {
#pragma unroll
                for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
                {
                    *reinterpret_cast<int4*>(&buffers[ii][params.local_rank * params.elts_total + iter_offset])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[iter_offset]);
                }
            }
            else
            {
                *reinterpret_cast<int4*>(&local_shared_buffer[iter_offset])
                    = *reinterpret_cast<int4 const*>(&local_input_buffer[iter_offset]);
            }
        }

        block_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);
    }
    else
    {
        multi_gpu_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    }

    for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * PACKED_ELTS)
    {
        PackedStruct vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            if constexpr (PUSH_MODE)
            {
                vals[ii].packed
                    = *reinterpret_cast<int4 const*>(&buffers[params.local_rank][ii * params.elts_total + iter_offset]);
            }
            else
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][iter_offset]);
            }
        }

        PackedStruct sums;
        sums.packed = {0, 0, 0, 0};
#pragma unroll
        for (int rank = 0; rank < RANKS_PER_NODE; ++rank)
        {
            int ii = (rank + RANKS_PER_NODE - params.local_rank) % RANKS_PER_NODE;
            sums.packed = add128b(sums, vals[ii]);
        }
        *reinterpret_cast<int4*>(&local_output_buffer[iter_offset]) = sums.packed;
    }
}

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true, bool PUSH_MODE = false, bool Bias = false,
    bool Residual = false>
static __global__ void __launch_bounds__(512, 1) twoShotAllReduceKernel(AllReduceParams params)
{

    int const bidx = blockIdx.x;
    int const tidx = threadIdx.x;
    int const grid_size = gridDim.x;

    static constexpr int PACKED_ELTS = 16 / sizeof(T);
    using PackedType = typename PackedOn16Bytes<T>::Type;

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);

    size_t const chunk_start = bidx * params.elts_per_block + tidx * PACKED_ELTS;
    size_t const chunk_end = min(chunk_start + params.elts_per_block, params.elts_per_rank);

    T* buffers[RANKS_PER_NODE];
    int ranks[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        ranks[ii] = rank;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    if constexpr (PUSH_MODE || COPY_INPUT)
    {
        for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
        {
#pragma unroll
            for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
            {
                size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
                if (offset_rank >= params.elts_total)
                {
                    continue;
                }

                if constexpr (PUSH_MODE)
                {
                    *reinterpret_cast<int4*>(&buffers[ii][params.local_rank * params.elts_per_rank + local_offset])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[offset_rank]);
                }
                else
                {
                    *reinterpret_cast<int4*>(&local_shared_buffer[offset_rank])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[offset_rank]);
                }
            }
        }
        block_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);
    }
    else
    {
        multi_gpu_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    }

    for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
    {
        size_t const responsible_block_offset = local_offset + params.rank_offset;

        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            if constexpr (PUSH_MODE)
            {
                vals[ii].packed
                    = *reinterpret_cast<int4 const*>(&local_shared_buffer[ii * params.elts_per_rank + local_offset]);
            }
            else
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][responsible_block_offset]);
            }
        }

        PackedType sums;
        sums.packed = {0, 0, 0, 0};
#pragma unroll
        for (int rank = 0; rank < RANKS_PER_NODE; ++rank)
        {
            int ii = (rank + RANKS_PER_NODE - params.local_rank) % RANKS_PER_NODE;
            sums.packed = add128b(sums, vals[ii]);
        }

        if constexpr (PUSH_MODE)
        {
            *reinterpret_cast<int4*>(&local_shared_buffer[local_offset]) = sums.packed;
        }
        else
        {
            *reinterpret_cast<int4*>(&local_shared_buffer[responsible_block_offset]) = sums.packed;
        }
    }

    block_barrier(
        params.peer_barrier_ptrs_out, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);

    for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
    {
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
            if (offset_rank >= params.elts_total)
            {
                continue;
            }
            PackedType sums, residual_vec, bias_vec;
            if constexpr (Bias)
            {
                bias_vec.packed
                    = *reinterpret_cast<int4 const*>(reinterpret_cast<T const*>(params.fusion_params.bias_buffer)
                        + offset_rank % params.fusion_params.hidden_size);
            }
            if constexpr (Residual)
            {
                residual_vec.packed = *reinterpret_cast<int4 const*>(
                    reinterpret_cast<T const*>(params.fusion_params.residual_buffer) + offset_rank);
            }
            if constexpr (PUSH_MODE)
            {
                *reinterpret_cast<int4*>(&local_output_buffer[offset_rank])
                    = *reinterpret_cast<int4*>(&buffers[ii][local_offset]);
                sums.packed = *reinterpret_cast<int4*>(&buffers[ii][local_offset]);
            }
            else
            {
                *reinterpret_cast<int4*>(&local_output_buffer[offset_rank])
                    = *reinterpret_cast<int4*>(&buffers[ii][offset_rank]);
                sums.packed = *reinterpret_cast<int4*>(&buffers[ii][offset_rank]);
            }
            if constexpr (Bias)
            {
                sums.packed = add128b(sums, bias_vec);
            }
            if constexpr (Residual)
            {
                sums.packed = add128b(sums, residual_vec);
            }
            *reinterpret_cast<int4*>(&local_output_buffer[offset_rank]) = sums.packed;
        }
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

bool configurationSupported(AllReduceStrategyType algo, size_t msg_size, size_t n_ranks, nvinfer1::DataType type)
{
    size_t elts_per_thread = 16 / common::getDTypeSize(type);
    int const msg_align = (algo == AllReduceStrategyType::TWOSHOT) ? n_ranks * elts_per_thread : elts_per_thread;
    bool supported_algo = (algo == AllReduceStrategyType::ONESHOT || algo == AllReduceStrategyType::TWOSHOT);
    return supported_algo && (msg_size % msg_align == 0);
}

std::tuple<int, int> kernelLaunchConfig(AllReduceStrategyType algo, AllReduceParams& params, size_t elts_per_thread)
{
    int blocks_per_grid = 1, threads_per_block = DEFAULT_BLOCK_SIZE;

    switch (algo)
    {
    case AllReduceStrategyType::ONESHOT:
    {
        CHECK(params.elts_total % elts_per_thread == 0);
        size_t const total_threads = roundUp(params.elts_total / elts_per_thread, WARP_SIZE);
        threads_per_block = std::min(DEFAULT_BLOCK_SIZE, total_threads);
        blocks_per_grid = std::min(static_cast<size_t>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
        params.elts_per_block = roundUp(divUp(params.elts_total, blocks_per_grid), elts_per_thread);
        break;
    }
    case AllReduceStrategyType::TWOSHOT:
    {
        CHECK(params.elts_total % (elts_per_thread * params.ranks_per_node) == 0);
        size_t const total_threads = roundUp(params.elts_total / (elts_per_thread * params.ranks_per_node), WARP_SIZE);

        while (total_threads % blocks_per_grid != 0 || total_threads / blocks_per_grid > DEFAULT_BLOCK_SIZE)
        {
            blocks_per_grid += 1;
        }

        threads_per_block = total_threads / blocks_per_grid;

        if (blocks_per_grid > MAX_ALL_REDUCE_BLOCKS)
        {
            size_t iter_factor = 1;
            while (blocks_per_grid / iter_factor > MAX_ALL_REDUCE_BLOCKS || blocks_per_grid % iter_factor)
            {
                iter_factor += 1;
            }
            blocks_per_grid /= iter_factor;
        }
        params.elts_per_rank = params.elts_total / params.ranks_per_node;
        params.rank_offset = params.local_rank * params.elts_per_rank;
        params.elts_per_block = roundUp(divUp(params.elts_per_rank, blocks_per_grid), elts_per_thread);
        break;
    }
    default: THROW("Algorithm not supported here.");
    }

    return std::make_tuple(blocks_per_grid, threads_per_block);
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false, bool Bias = false,
    bool Affine = false>
void AllReduceNormKernelLaunch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    CHECK_WITH_INFO(
        (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM || fusionOp == AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM),
        "Unsupported AllReduceFusionOp: %d", static_cast<int>(fusionOp));
    if (algo == AllReduceStrategyType::ONESHOT)
    {
        reduce_fusion::one_shot_all_reduce_norm_kernel_launcher<T, RANKS_PER_NODE, Bias, Affine>(
            params, stream, fusionOp);
    }
    else
    {
        CHECK_WITH_INFO(!(USE_MEMCPY && PUSH_MODE), "Memcpy cannot be used with PUSH_MODE.");
        size_t elts_per_thread = 16 / sizeof(T);
        auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(algo, params, elts_per_thread);
        if (USE_MEMCPY)
        {
            hipMemcpyAsync(params.peer_comm_buffer_ptrs[params.local_rank], params.local_input_buffer_ptr,
                params.elts_total * sizeof(T), hipMemcpyDeviceToDevice, stream);
        }
        auto output_ptr = params.local_output_buffer_ptr;
        params.local_output_buffer_ptr = params.fusion_params.intermediate_buffer;

        if (sugesstify::common::getEnvEnablePDL())
        {
            LOG_DEBUG("Enable PDL in twoShotAllReduceKernel");
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = blocks_per_grid;
            kernelConfig.blockDim = threads_per_block;
            kernelConfig.dynamicSmemBytes = 0;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            CUDA_CHECK(cudaLaunchKernelEx(
                &kernelConfig, twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE, Bias, true>, params));
        }
        else
        {
            twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE, Bias, true>
                <<<blocks_per_grid, threads_per_block, 0, stream>>>(params);
        }
        params.local_output_buffer_ptr = output_ptr;
        reduce_fusion::rms_norm_kernel_launcher<T, false, false, Affine>(params, stream, fusionOp);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceNormDispatch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    if (params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, true, true>(
            algo, config, fusionOp, params, stream);
    }
    else if (params.fusion_params.bias_buffer && !params.fusion_params.weight_buffer)
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, true, false>(
            algo, config, fusionOp, params, stream);
    }
    else if (!params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, false, true>(
            algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, false, false>(
            algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceDispatch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    CHECK(fusionOp == AllReduceFusionOp::NONE);
    CHECK_WITH_INFO(!(USE_MEMCPY && PUSH_MODE), "Memcpy cannot be used with PUSH_MODE.");
    size_t elts_per_thread = 16 / sizeof(T);
    auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(algo, params, elts_per_thread);
    if (USE_MEMCPY)
    {
        hipMemcpyAsync(params.peer_comm_buffer_ptrs[params.local_rank], params.local_input_buffer_ptr,
            params.elts_total * sizeof(T), hipMemcpyDeviceToDevice, stream);
    }
    if (algo == AllReduceStrategyType::ONESHOT)
    {
        oneShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE>
            <<<blocks_per_grid, threads_per_block, 0, stream>>>(params);
    }
    else
    {
        twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE>
            <<<blocks_per_grid, threads_per_block, 0, stream>>>(params);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceDispatchMemcpy(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    if (fusionOp == AllReduceFusionOp::NONE)
    {
        LOG_DEBUG("AllReduceDispatch enabled");
        AllReduceDispatch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY>(algo, config, fusionOp, params, stream);
    }
    else
    {
        LOG_DEBUG("AllReduceNormDispatch enabled");
        AllReduceNormDispatch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY>(algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false>
void AllReduceDispatchPushMode(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    if (static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(config)
        & static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(AllReduceStrategyConfig::USE_MEMCPY))
    {
        AllReduceDispatchMemcpy<T, RANKS_PER_NODE, PUSH_MODE, true>(algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceDispatchMemcpy<T, RANKS_PER_NODE, PUSH_MODE, false>(algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE>
void AllReduceDispatchRanksPerNode(AllReduceStrategyType algo, AllReduceStrategyConfig config,
    AllReduceFusionOp fusionOp, AllReduceParams& params, hipStream_t stream)
{
    if (static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(config)
        & static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(AllReduceStrategyConfig::PUSH_MODE))
    {
        AllReduceDispatchPushMode<T, RANKS_PER_NODE, true>(algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceDispatchPushMode<T, RANKS_PER_NODE, false>(algo, config, fusionOp, params, stream);
    }
}

template <typename T>
void AllReduceDispatchType(AllReduceParams& params, AllReduceStrategyType strat, AllReduceStrategyConfig config,
    AllReduceFusionOp fusionOp, hipStream_t stream)
{
    switch (params.ranks_per_node)
    {
    case 2: AllReduceDispatchRanksPerNode<T, 2>(strat, config, fusionOp, params, stream); break;
    case 4: AllReduceDispatchRanksPerNode<T, 4>(strat, config, fusionOp, params, stream); break;
    case 6: AllReduceDispatchRanksPerNode<T, 6>(strat, config, fusionOp, params, stream); break;
    case 8: AllReduceDispatchRanksPerNode<T, 8>(strat, config, fusionOp, params, stream); break;
    default: THROW("Custom all reduce only supported on {2, 4, 6, 8} GPUs per node.");
    }
}

AllReduceParams AllReduceParams::deserialize(int64_t* buffer, size_t tpSize, size_t tpRank, nvinfer1::DataType dataType,
    int token_num, int hidden_size, AllReduceFusionOp op)
{
    void* const* buffer_ptrs = reinterpret_cast<void* const*>(buffer);
    int flag_offset;
    if (op == AllReduceFusionOp::RESIDUAL_RMS_NORM
        && reduce_fusion::is_lamport_supported(dataType, token_num, hidden_size))
    {
        flag_offset = 0;
    }
    else
    {
        flag_offset = 1;
    }
    auto const flag_ptr
        = &buffer[sugesstify::utils::customAllReduceUtils::NUM_POINTERS_PER_RANK * tpSize + flag_offset];
    *flag_ptr += 1;
    LOG_TRACE("AllReduceParams's flag value is %d, flag offset %d", *flag_ptr, flag_offset);
    uint32_t flag_value = *flag_ptr;
    AllReduceParams params;
    auto const buffer_offset = (flag_value % 2 == 0) ? 0 : tpSize;

    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_comm_buffer_ptrs[i] = buffer_ptrs[buffer_offset + i];
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_in[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[2 * tpSize + i]);
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_out[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[3 * tpSize + i]);
    }
    params.barrier_flag = flag_value;
    params.ranks_per_node = tpSize;
    params.local_rank = tpRank;

    return params;
}

void customAllReduce(kernels::AllReduceParams& params, nvinfer1::DataType dataType, AllReduceStrategyType strat,
    AllReduceStrategyConfig config, AllReduceFusionOp fusionOp, hipStream_t stream)
{
    CHECK_WITH_INFO(configurationSupported(strat, params.elts_total, params.ranks_per_node, dataType),
        "Custom all-reduce configuration unsupported");

    sync_check_cuda_error();

    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT: AllReduceDispatchType<float>(params, strat, config, fusionOp, stream); break;
    case nvinfer1::DataType::kHALF: AllReduceDispatchType<half>(params, strat, config, fusionOp, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        AllReduceDispatchType<__hip_bfloat16>(params, strat, config, fusionOp, stream);
        break;
#endif
    default: THROW("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error();
}

template <typename T>
void launchResidualRmsNormKernel(kernels::AllReduceParams& params, hipStream_t stream, AllReduceFusionOp fusionOp)
{
    if (params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        reduce_fusion::rms_norm_kernel_launcher<T, true, true, true>(params, stream, fusionOp);
    }
    else if (params.fusion_params.bias_buffer && !params.fusion_params.weight_buffer)
    {
        reduce_fusion::rms_norm_kernel_launcher<T, true, true, false>(params, stream, fusionOp);
    }
    else if (!params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        reduce_fusion::rms_norm_kernel_launcher<T, false, true, true>(params, stream, fusionOp);
    }
    else
    {
        reduce_fusion::rms_norm_kernel_launcher<T, false, true, false>(params, stream, fusionOp);
    }
}

void residualRmsNorm(
    kernels::AllReduceParams& params, nvinfer1::DataType dataType, hipStream_t stream, AllReduceFusionOp fusionOp)
{
    sync_check_cuda_error();
    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT: launchResidualRmsNormKernel<float>(params, stream, fusionOp); break;
    case nvinfer1::DataType::kHALF: launchResidualRmsNormKernel<half>(params, stream, fusionOp); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: launchResidualRmsNormKernel<__hip_bfloat16>(params, stream, fusionOp); break;
#endif
    default: THROW("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error();
}

void lamportInitialize(void* buffer, size_t size, nvinfer1::DataType dataType, hipStream_t stream)
{
    sync_check_cuda_error();
    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT:
        reduce_fusion::lamport_initialize_kernel_launcher<float>(buffer, size, stream);
        break;
    case nvinfer1::DataType::kHALF:
        reduce_fusion::lamport_initialize_kernel_launcher<half>(buffer, size, stream);
        break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        reduce_fusion::lamport_initialize_kernel_launcher<__hip_bfloat16>(buffer, size, stream);
        break;
#endif
    default: THROW("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error();
}

}
