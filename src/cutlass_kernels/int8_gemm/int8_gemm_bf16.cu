#include "hip/hip_runtime.h"

#include "../src/cutlass_kernels/int8_gemm/int8_gemm_template.h"

namespace suggestify
{
namespace kernels
{
namespace cutlass_kernels
{

#ifdef ENABLE_BF16
template class CutlassInt8GemmRunner<__hip_bfloat16>;
#endif

} // namespace cutlass_kernels
} // namespace kernels
} // namespace suggestify
