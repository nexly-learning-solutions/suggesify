#include "hip/hip_runtime.h"


#include "../src/cutlass_kernels/fpA_intB_gemm/fpA_intB_gemm_template.h"

namespace suggestify
{
namespace kernels
{
namespace cutlass_kernels
{
#ifdef ENABLE_FP8
template class CutlassFpAIntBGemmRunner<__hip_fp8_e4m3_fnuz,             /*Activation Type*/
    cutlass::uint4b_t,                                             /*Weight Type*/
    cutlass::WeightOnlyQuantOp::FINEGRAINED_SCALE_AND_ZEROS, half, /*Scale and Zero Type*/
    __hip_bfloat16,                                                 /*Bias type Type*/
    __hip_bfloat16                                                  /*Output type Type*/
    >;
#endif
} // namespace cutlass_kernels
} // namespace kernels
} // namespace suggestify
