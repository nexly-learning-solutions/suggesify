

#include "../decoderMaskedMultiheadAttentionLaunch.h"

namespace suggestify
{
namespace kernels
{

namespace
{
auto constexpr kSizePerHead = 64;
} // namespace

namespace mmha
{

INSTANTIATE_MMHA_LAUNCHERS_WITH_IMPLICIT_REL_ATTN_BIAS(float, kSizePerHead)

} // namespace mmha

} // namespace kernels
} // namespace suggestify
