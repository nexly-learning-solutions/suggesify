

#include "../decoderMaskedMultiheadAttentionLaunch.h"

namespace suggestify
{
namespace kernels
{

namespace
{
auto constexpr kSizePerHead = 112;
} // namespace

namespace mmha
{

#ifndef FAST_BUILD // skip mmha_112 for fast build
INSTANTIATE_MMHA_LAUNCHERS(float, kSizePerHead)
#endif             // FAST_BUILD

} // namespace mmha

} // namespace kernels
} // namespace suggestify
