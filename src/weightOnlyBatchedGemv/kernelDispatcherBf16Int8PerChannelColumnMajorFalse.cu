

#include "../src/weightOnlyBatchedGemv/kernelDispatcher.h"

namespace sugesstify
{
namespace kernels
{
namespace weight_only
{
INSTANTIATE_WEIGHT_ONLY_CUDA_DISPATCHERS(
    KernelType::BF16Int8PerChannel, BF16DetailsA, Int8DetailsW, ColumnMajor, false, 64);
} // namespace weight_only
} // namespace kernels
} // namespace sugesstify
