#include "hip/hip_runtime.h"


#include "unfusedAttentionKernels_2_template.h"

namespace suggestify
{
namespace kernels
{

#ifdef ENABLE_BF16
INSTANTIATE_ATTENTION_INPUT_OUTPUT_PROCESSING(__hip_bfloat16, __hip_bfloat16, KVBlockArray);
INSTANTIATE_ATTENTION_INPUT_OUTPUT_PROCESSING(__hip_bfloat16, __hip_bfloat16, KVLinearBuffer);
#endif

} // namespace kernels
} // namespace suggestify
