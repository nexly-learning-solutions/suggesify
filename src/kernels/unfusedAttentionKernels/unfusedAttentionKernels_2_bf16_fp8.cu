#include "hip/hip_runtime.h"


#include "unfusedAttentionKernels_2_template.h"

namespace suggestify
{
namespace kernels
{

#ifdef ENABLE_BF16
INSTANTIATE_ATTENTION_INPUT_OUTPUT_PROCESSING(__hip_bfloat16, __hip_fp8_e4m3_fnuz, KVBlockArray);
INSTANTIATE_ATTENTION_INPUT_OUTPUT_PROCESSING(__hip_bfloat16, __hip_fp8_e4m3_fnuz, KVLinearBuffer);
#endif

} // namespace kernels
} // namespace suggestify
