#include "hip/hip_runtime.h"


#include "../decoderMaskedMultiheadAttentionLaunch.h"

namespace suggestify
{
namespace kernels
{

namespace
{
auto constexpr kSizePerHead = 128;
} // namespace

namespace mmha
{

#ifdef ENABLE_BF16
INSTANTIATE_MMHA_LAUNCHERS_WITH_BLOCK_SPARSE_ATTN(__hip_bfloat16, kSizePerHead)
#endif

} // namespace mmha

} // namespace kernels
} // namespace suggestify
