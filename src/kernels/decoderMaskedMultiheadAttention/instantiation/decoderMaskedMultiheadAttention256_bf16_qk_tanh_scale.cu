#include "hip/hip_runtime.h"


#include "../decoderMaskedMultiheadAttentionLaunch.h"

namespace suggestify
{
namespace kernels
{

namespace
{
auto constexpr kSizePerHead = 256;
} // namespace

namespace mmha
{

#ifdef ENABLE_BF16
INSTANTIATE_MMHA_LAUNCHERS_WITH_ATTN_LOGIT_SOFTCAPPING_SCALE(__hip_bfloat16, kSizePerHead)
#endif

} // namespace mmha

} // namespace kernels
} // namespace suggestify
