#include "hip/hip_runtime.h"


#include "../decoderMaskedMultiheadAttentionLaunch.h"

namespace suggestify
{
namespace kernels
{

namespace
{
auto constexpr kSizePerHead = 192;
} // namespace

namespace mmha
{

#ifndef FAST_BUILD // skip mmha_192 for fast build
#ifdef ENABLE_BF16
INSTANTIATE_MMHA_LAUNCHERS(__hip_bfloat16, kSizePerHead)
#endif // ENABLE_BF16
#endif // FAST_BUILD

} // namespace mmha

} // namespace kernels
} // namespace suggestify
