

#include "../decoderMaskedMultiheadAttentionLaunch.h"

namespace suggestify
{
namespace kernels
{

namespace
{
auto constexpr kSizePerHead = 128;
} // namespace

namespace mmha
{

INSTANTIATE_MMHA_LAUNCHERS_WITH_ATTN_LOGIT_SOFTCAPPING_SCALE(float, kSizePerHead)

} // namespace mmha

} // namespace kernels
} // namespace suggestify
