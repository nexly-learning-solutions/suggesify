#include "hip/hip_runtime.h"


#include "../decoderMaskedMultiheadAttentionLaunch.h"

namespace suggestify
{
namespace kernels
{

namespace
{
auto constexpr kSizePerHead = 96;
} // namespace

namespace mmha
{

#ifndef FAST_BUILD // skip mmha_96 for fast build
#ifdef ENABLE_BF16
INSTANTIATE_MMHA_LAUNCHERS(__hip_bfloat16, kSizePerHead)
#endif // ENABLE_BF16
#endif // FAST_BUILD

} // namespace mmha

} // namespace kernels
} // namespace suggestify
