#include "hip/hip_runtime.h"


#include "../decoderMaskedMultiheadAttentionLaunch.h"

namespace suggestify
{
namespace kernels
{

namespace
{
auto constexpr kSizePerHead = 112;
} // namespace

namespace mmha
{

#ifndef FAST_BUILD // skip mmha_112 for fast build
#ifdef ENABLE_BF16
INSTANTIATE_MMHA_LAUNCHERS(__hip_bfloat16, kSizePerHead)
#endif // ENABLE_BF16
#endif // FAST_BUILD

} // namespace mmha

} // namespace kernels
} // namespace suggestify
