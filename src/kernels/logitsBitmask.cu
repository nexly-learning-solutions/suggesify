#include "hip/hip_runtime.h"


#include "../common/cudaUtils.h"
#include "../src/logitsBitmask.h"

using namespace suggestify::common;
using namespace suggestify::runtime;

namespace suggestify
{
namespace kernels
{

constexpr int32_t kBitsPerMaskElement = 32;
constexpr int32_t kThreadsPerBlock = 512;

template <typename T>
__device__ T GetNegativeInfinity()
{
    return -INFINITY;
}

template <>
__device__ half GetNegativeInfinity<half>()
{
    return __float2half(-INFINITY);
}

template <typename T>
__global__ void __launch_bounds__(512) logitsBitmaskKernel(
    T** __restrict__ logits, uint32_t const** __restrict__ bitmask, int32_t vocabSizePadded, int32_t bitmaskSize)
{
    int batchIdx = blockIdx.y;
    int bitmaskIdx = blockDim.x * blockIdx.x + threadIdx.x;
    if (bitmaskIdx >= bitmaskSize)
    {
        return;
    }

    uint32_t bitmaskVal = bitmask[batchIdx][bitmaskIdx];
    T* logitsPtr = logits[batchIdx] + bitmaskIdx * kBitsPerMaskElement;
    for (int i = 0; i < kBitsPerMaskElement; ++i)
    {
        if (bitmaskIdx * kBitsPerMaskElement + i >= vocabSizePadded)
        {
            break;
        }
        if (!(bitmaskVal & 1))
        {
            // TODO(enweiz): Fix uncoalesced global memory access here.
            logitsPtr[i] = GetNegativeInfinity<T>();
        }
        bitmaskVal >>= 1;
    }
}

template <typename T>
void invokeLogitsBitmask(
    T** logits, uint32_t const** bitmask, int32_t batchSize, int32_t vocabSizePadded, hipStream_t stream)
{
    int bitmaskSize = ceilDiv(vocabSizePadded, kBitsPerMaskElement);
    dim3 grid(ceilDiv(bitmaskSize, kThreadsPerBlock), batchSize);
    dim3 block(kThreadsPerBlock);

    logitsBitmaskKernel<T><<<grid, block, 0, stream>>>(logits, bitmask, vocabSizePadded, bitmaskSize);
}

template void invokeLogitsBitmask<float>(
    float** logits, uint32_t const** bitmask, int32_t batchSize, int32_t vocabSizePadded, hipStream_t stream);
template void invokeLogitsBitmask<half>(
    half** logits, uint32_t const** bitmask, int32_t batchSize, int32_t vocabSizePadded, hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeLogitsBitmask<__hip_bfloat16>(
    __hip_bfloat16** logits, uint32_t const** bitmask, int32_t batchSize, int32_t vocabSizePadded, hipStream_t stream);
#endif
} // namespace kernels
} // namespace suggestify
