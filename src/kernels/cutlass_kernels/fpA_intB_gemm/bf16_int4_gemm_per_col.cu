#include "hip/hip_runtime.h"


#include "../src/cutlass_kernels/fpA_intB_gemm/fpA_intB_gemm_template.h"

namespace suggestify
{
namespace kernels
{
namespace cutlass_kernels
{
#ifdef ENABLE_BF16
template class CutlassFpAIntBGemmRunner<__hip_bfloat16, cutlass::uint4b_t,
    cutlass::WeightOnlyQuantOp::PER_COLUMN_SCALE_ONLY>;
#endif
} // namespace cutlass_kernels
} // namespace kernels
} // namespace suggestify
