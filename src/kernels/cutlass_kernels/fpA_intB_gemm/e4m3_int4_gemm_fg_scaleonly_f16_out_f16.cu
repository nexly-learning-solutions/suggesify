#include "hip/hip_runtime.h"

#include "../src/cutlass_kernels/fpA_intB_gemm/fpA_intB_gemm_template.h"

namespace suggestify
{
namespace kernels
{
namespace cutlass_kernels
{
#ifdef ENABLE_FP8
template class CutlassFpAIntBGemmRunner<__hip_fp8_e4m3_fnuz,        /*Activation Type*/
    cutlass::uint4b_t,                                        /*Weight Type*/
    cutlass::WeightOnlyQuantOp::FINEGRAINED_SCALE_ONLY, half, /*Scale and Zero Type*/
    half,                                                     /*Bias type Type*/
    half                                                      /*Output type Type*/
    >;
#endif
} // namespace cutlass_kernels
} // namespace kernels
} // namespace suggestify
