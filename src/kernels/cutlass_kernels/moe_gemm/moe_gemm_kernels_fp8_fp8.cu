#include "hip/hip_runtime.h"


#include "../src/cutlass_kernels/moe_gemm/moe_gemm_kernels_template.h"

namespace suggestify
{
#ifdef ENABLE_FP8
template class MoeGemmRunner<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half>;
#ifdef ENABLE_BF16
template class MoeGemmRunner<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16>;
#endif
// template class MoeGemmRunner<__hip_fp8_e5m2_fnuz, __hip_fp8_e5m2_fnuz>;
#endif
} // namespace suggestify
