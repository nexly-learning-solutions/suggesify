#include "hip/hip_runtime.h"


#include "../src/cutlass_kernels/moe_gemm/moe_gemm_kernels_template.h"

namespace suggestify
{
#ifdef ENABLE_BF16
template class MoeGemmRunner<__hip_bfloat16, uint8_t, __hip_bfloat16>;
#endif
} // namespace suggestify
