#include "hip/hip_runtime.h"

#include "../common/cudaTypeUtils.cuh"
#include "../common/quantTypeUtils.cuh"
#include "../common/reduceKernelUtils.cuh"
#include "rmsnormKernels.h"

using namespace sugesstify::common;

namespace sugesstify
{
namespace kernels
{

template <typename Tf, typename T>
__inline__ __device__ Tf compute_rmsnorm(Tf val, float s_variance, T const* gamma, T const* beta, int i)
{
    Tf ret = val * s_variance * cuda_cast<Tf>(gamma[i]);
    if (beta != nullptr)
    {
        ret = ret + cuda_cast<Tf>(beta[i]);
    }
    return ret;
}

template <typename T, typename QuantT, bool USE_SHMEM>
__global__ void generalRmsNorm(T const* input, T const* gamma, T const* beta, T* normed_output, float const eps,
    int tokens, int hidden_dim, float const* clampPtr, float const* scale_orig_quant_per_tensor,
    float* scale_orig_quant_per_token, float* sum_per_token, QuantT* normed_output_quant, bool hasFp8MinScaling)
{
    constexpr auto num_elems_T = num_elems<T>::value;
    using QuantT_packed_t = typename packed_as<QuantT, num_elems_T>::type;
    using float_packed_t = typename packed_as<float, num_elems_T>::type;
    using T_scalar = typename packed_as<T, 1>::type;

    T const clampMin = cuda_cast<T>(clampPtr ? clampPtr[0] : -FLT_MAX);
    T const clampMax = cuda_cast<T>(clampPtr ? clampPtr[1] : FLT_MAX);

    static constexpr float MAX_QUANT_VAL = QuantTypeStaticVals<QuantT>::MAX_VAL;
    static constexpr float MIN_SCALING_FACTOR = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR;
    static constexpr float MIN_SCALING_FACTOR_RCP = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR_RCP;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T* shmem = reinterpret_cast<T*>(_shmem);

    __shared__ float s_variance;

    int const tidx = threadIdx.x;
    int const bidx = blockIdx.x;

    float variance = 0.0f;
    float local_var_sum = 0.0f;

    int const n_elems = hidden_dim / num_elems_T;
    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        T const val = input[bidx * n_elems + i];
        if (USE_SHMEM)
        {
            shmem[i] = val;
        }

        float_packed_t const val_f = cuda_cast<float_packed_t>(val);

        local_var_sum += cuda_sum<float>(val_f * val_f);
    }

    float packed[1] = {local_var_sum};
    blockReduceSumV2<float, 1>(packed);
    variance = packed[0];

    if (threadIdx.x == 0)
    {
        variance = (variance / hidden_dim);
        s_variance = rsqrtf(variance + eps);
    }
    __syncthreads();

    bool const with_per_token_scaling = scale_orig_quant_per_token != nullptr;
    bool const with_per_tensor_scaling = scale_orig_quant_per_tensor != nullptr;
    bool const with_per_token_sum = sum_per_token != nullptr;

    float_packed_t const scale_orig_quant
        = cuda_cast<float_packed_t>(with_per_tensor_scaling ? *scale_orig_quant_per_tensor : 0.0f);
    T_scalar amax = 1e-6f;
    float local_sum = 0.f;

    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        int const index = bidx * n_elems + i;
        float_packed_t const val_f = cuda_cast<float_packed_t>(USE_SHMEM ? shmem[i] : input[index]);
        T val = cuda_cast<T>(compute_rmsnorm(val_f, s_variance, gamma, beta, i));

        if (with_per_token_scaling)
        {
            val = cuda_clamp(val, clampMin, clampMax);
            amax = cuda_max(cuda_max<T_scalar, T>(cuda_abs(val)), amax);
            if (USE_SHMEM)
            {
                shmem[i] = val;
            }
        }
        else if (with_per_tensor_scaling)
        {
            val = cuda_clamp(val, clampMin, clampMax);
            reinterpret_cast<QuantT_packed_t*>(normed_output_quant)[index]
                = cuda_cast<QuantT_packed_t>(cuda_cast<float_packed_t>(val) * scale_orig_quant);
        }
        else
        {
            normed_output[index] = val;
        }

        if (with_per_token_sum)
        {
            local_sum += cuda_sum<float>(cuda_cast<float_packed_t>(val));
        }
    }

    if (with_per_token_scaling)
    {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(amax));
        float const dynamic_per_token_scale
            = hasFp8MinScaling ? fminf(MAX_QUANT_VAL / abs_max_f, MIN_SCALING_FACTOR_RCP) : (MAX_QUANT_VAL / abs_max_f);
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            int const index = bidx * n_elems + i;
            float_packed_t val_f = cuda_cast<float_packed_t>(USE_SHMEM ? shmem[i] : input[index]);
            if (!USE_SHMEM)
            {
                val_f = compute_rmsnorm(val_f, s_variance, gamma, beta, i);
            }

            reinterpret_cast<QuantT_packed_t*>(normed_output_quant)[index]
                = cuda_cast<QuantT_packed_t>(val_f * cuda_cast<float_packed_t>(dynamic_per_token_scale));
        }
        if (tidx == 0)
        {
            scale_orig_quant_per_token[bidx] = hasFp8MinScaling
                ? cuda_max(abs_max_f / MAX_QUANT_VAL, MIN_SCALING_FACTOR)
                : abs_max_f / MAX_QUANT_VAL;
        }
    }

    if (with_per_token_sum)
    {
        float packed_sum[1] = {local_sum};
        blockReduceSumV2<float, 1>(packed_sum);
        if (tidx == 0)
        {
            sum_per_token[bidx] = packed_sum[0];
        }
    }
}

template <typename T, typename QuantT>
void dispatch_rmsnorm_type_square_method(T const* input, T const* gamma, T const* beta, T* normed_output,
    float const eps, int tokens, int hidden_dim, float const* clampPtr, float const* scale_orig_quant_per_tensor,
    float* scale_orig_quant_per_token, float* sum_per_token, QuantT* normed_output_quant, bool const hasFp8MinScaling,
    dim3 const grid, dim3 const block, size_t const shmem_size, hipStream_t stream)
{
    bool use_shmem = true;
    if (shmem_size >= (48 << 10))
    {
        hipError_t ret = hipFuncSetAttribute(reinterpret_cast<const void*>(
            generalRmsNorm<T), QuantT, true>, hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size);
        use_shmem = (ret == hipSuccess);
    }

    if (use_shmem)
    {
        generalRmsNorm<T, QuantT, true><<<grid, block, shmem_size, stream>>>(input, gamma, beta, normed_output, eps,
            tokens, hidden_dim, clampPtr, scale_orig_quant_per_tensor, scale_orig_quant_per_token, sum_per_token,
            normed_output_quant, hasFp8MinScaling);
    }
    else
    {
        generalRmsNorm<T, QuantT, false><<<grid, block, shmem_size, stream>>>(input, gamma, beta, normed_output, eps,
            tokens, hidden_dim, clampPtr, scale_orig_quant_per_tensor, scale_orig_quant_per_token, sum_per_token,
            normed_output_quant, hasFp8MinScaling);
    }
}

template <typename T, typename QuantT>
void dispatch_rmsnorm_type(T const* input, T const* gamma, T const* beta, T* normed_output, float const eps, int tokens,
    int hidden_dim, float const* clampPtr, float const* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token,
    float* sum_per_token, QuantT* normed_output_quant, bool const hasFp8MinScaling, dim3 const grid, dim3 const block,
    size_t const shmem_size, hipStream_t stream)
{
    dispatch_rmsnorm_type_square_method(input, gamma, beta, normed_output, eps, tokens, hidden_dim, clampPtr,
        scale_orig_quant_per_tensor, scale_orig_quant_per_token, sum_per_token, normed_output_quant, hasFp8MinScaling,
        grid, block, shmem_size, stream);
}

template <typename T, typename QuantT>
void invokeGeneralRmsNorm(T* out, T const* input, T const* gamma, T const* beta, float const eps, int const tokens,
    int const hidden_dim, QuantMode quantMode, hipStream_t stream, float const* clampPtr, float const* scale,
    float* dynamic_scale, float* sum_per_token, QuantT* normed_output_quant)
{
    dim3 grid(tokens);
    dim3 block(min(hidden_dim, 1024));
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size = 2;
    size_t const shmem_size = hidden_dim * sizeof(T);
    bool const use_vec_type = (hidden_dim % vec_size == 0)
        && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
            || std::is_same<T, __hip_bfloat16>::value
#endif
        );

    bool hasFp8MinScaling = quantMode.hasFp8RowWise();

    if (use_vec_type)
    {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_rmsnorm_type(reinterpret_cast<Tp const*>(input), reinterpret_cast<Tp const*>(gamma),
            reinterpret_cast<Tp const*>(beta), reinterpret_cast<Tp*>(out), eps, tokens, hidden_dim, clampPtr, scale,
            dynamic_scale, sum_per_token, normed_output_quant, hasFp8MinScaling, grid, block, shmem_size, stream);
    }
    else
    {
        dispatch_rmsnorm_type(input, gamma, beta, out, eps, tokens, hidden_dim, clampPtr, scale, dynamic_scale,
            sum_per_token, normed_output_quant, hasFp8MinScaling, grid, block, shmem_size, stream);
    }
}

#define INSTANTIATE_GENERAL_RMSNORM(T, QuantT)                                                                         \
    template void invokeGeneralRmsNorm(T* out, const T* input, const T* gamma, const T* beta, const float eps,         \
        const int tokens, const int hidden_dim, QuantMode quantMode, hipStream_t stream, float const* clampPtr,       \
        const float* scale, float* dynamic_scale, float* sum_per_token, QuantT* normed_output_quant);

INSTANTIATE_GENERAL_RMSNORM(float, int8_t);
INSTANTIATE_GENERAL_RMSNORM(half, int8_t);

#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_RMSNORM(__hip_bfloat16, int8_t);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_GENERAL_RMSNORM(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_GENERAL_RMSNORM(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_RMSNORM(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

}
}
