#include "hip/hip_runtime.h"


#include "userbuffers.h"
#include "utils.h"

namespace sugesstify::kernels::ub
{
using namespace sugesstify::runtime::ub;
#define MAX_THREADS 1024
#define TIMEOUT 200000000000ull

__forceinline__ __device__ void multi_gpu_block_barrier(int reduce_id, int volatile* flag)
{
#ifdef UB_TIMEOUT_ENABLED
    clock_t s = clock64();
#endif
    while (*flag < reduce_id)
    {
#ifdef UB_TIMEOUT_ENABLED
        if (clock64() - s > 2ull * TIMEOUT)
        {
            printf("NVONLY RSBAR:SM %d [%d]:expecting %d got %d\n", blockIdx.x, threadIdx.x, reduce_id, *flag);
            break;
        }
#endif
    }
}

template <typename DType, int RANKS>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_rw(int const op, int const flagoffset, int const firstrank, int const myrank,
        int const gpustep, size_t const lineoffset, int const numlines, void** commbuff, int const handleridx)
{
#if __CUDA_ARCH__ >= 900
    cudaTriggerProgrammaticLaunchCompletion();
#endif
    __shared__ int4* userptr[RANKS];
    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = (*reduceidptr) + 1;
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
#if __CUDA_ARCH__ >= 900
        cudaGridDependencySynchronize();
#endif
        flagptr[physgpu] = reduce_id;
        userptr[threadIdx.x] = reinterpret_cast<int4*>(commbuff[targetgpu + handleridx]);
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
        reduce_id++;
    }
    __syncthreads();

    int warp = blockIdx.x + (threadIdx.x >> 5);
    int dest[RANKS];
#pragma unroll
    for (int i = 0; i < RANKS; i++)
        dest[i] = (i + myrank + warp) & (RANKS - 1);

    __syncthreads();
    for (int line = threadIdx.x + blockDim.x * (myrank + RANKS * blockIdx.x); line < numlines;
         line += blockDim.x * gridDim.x * RANKS)
    {
        int4 val[RANKS];

#pragma unroll
        for (int i = 0; i < RANKS; i++)
        {
            val[i] = userptr[dest[i]][lineoffset + line];
        }

        int4 sum = val[0];
        DType* s = reinterpret_cast<DType*>(&sum);

#pragma unroll
        for (int i = 1; i < RANKS; i++)
        {
            DType* x = reinterpret_cast<DType*>(&val[i]);
#pragma unroll
            for (int j = 0; j < 8; j++)
                s[j] += x[j];
        }
#pragma unroll
        for (int i = 0; i < RANKS; i++)
        {
            userptr[dest[i]][lineoffset + line] = sum;
        }
    }

    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence_system();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // fp16 inplace reduce kernel (Volta,Hopper)

template <typename DType, int RANKS>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_rr(int const op, int const flagoffset, int const firstrank, int const myrank,
        int const gpustep, size_t const lineoffset, int const numlines, void** commbuff, int const handleridx)
{
#if __CUDA_ARCH__ >= 900
    cudaTriggerProgrammaticLaunchCompletion();
#endif
    __shared__ int4* userptr[RANKS];
    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = (*reduceidptr) + 1;
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
#if __CUDA_ARCH__ >= 900
        cudaGridDependencySynchronize();
#endif
        flagptr[physgpu] = reduce_id;
        userptr[threadIdx.x] = reinterpret_cast<int4*>(commbuff[targetgpu + handleridx]);
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
        reduce_id++;
    }
    __syncthreads();

    int warp = blockIdx.x + (threadIdx.x >> 5);
    int dest[RANKS];
#pragma unroll
    for (int i = 0; i < RANKS; i++)
        dest[i] = (i + myrank + warp) & (RANKS - 1);

    __syncthreads();
    for (int line = threadIdx.x + blockDim.x * (myrank + RANKS * blockIdx.x); line < numlines;
         line += blockDim.x * gridDim.x * RANKS)
    {
        int4 val[RANKS];

#pragma unroll
        for (int i = 0; i < RANKS; i++)
        {
            val[i] = userptr[dest[i]][lineoffset + line];
        }

        int4 sum = val[0];
        DType* s = reinterpret_cast<DType*>(&sum);

#pragma unroll
        for (int i = 1; i < RANKS; i++)
        {
            DType* x = reinterpret_cast<DType*>(&val[i]);
#pragma unroll
            for (int j = 0; j < 8; j++)
                s[j] += x[j];
        }

        userptr[myrank][lineoffset + line] = sum;
    }
    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }

    int skipmy = 0;
#pragma unroll
    for (int i = 0; i < RANKS; i++)
    {
        int dst = (i + warp + myrank) & (RANKS - 1);
        if (dst == myrank)
        {
            skipmy++;
            continue;
        }
        dest[i - skipmy] = dst;
    }
    __syncthreads();

    for (int line = threadIdx.x + blockDim.x * RANKS * blockIdx.x; line < numlines;
         line += blockDim.x * gridDim.x * RANKS)
    {
        int4 val[RANKS - 1];

#pragma unroll
        for (int i = 0; i < RANKS - 1; i++)
        {
            val[i] = userptr[dest[i]][lineoffset + line + blockDim.x * dest[i]];
        }

#pragma unroll
        for (int i = 0; i < RANKS - 1; i++)
        {
            userptr[myrank][lineoffset + line + blockDim.x * dest[i]] = val[i];
        }
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // fp16 inplace reduce kernel (Ampere)

#if __CUDA_ARCH__ >= 900
template <typename ValType, typename PtrType>
__device__ __forceinline__ void MULTIMEM_ST(ValType val, PtrType ptr)
{
    asm volatile(
        "multimem.st.global.v4.f32 [%0], {%1,%2,%3,%4};" ::"l"(ptr), "r"(val.x), "r"(val.y), "r"(val.z), "r"(val.w)
        : "memory");
}

template <typename ValType, typename PtrType>
__device__ __forceinline__ void MULTIMEM_ST2(ValType& val, PtrType ptr)
{
    asm volatile("multimem.st.global.v2.f32 [%0], {%1,%2};" ::"l"(ptr), "r"(val.x), "r"(val.y) : "memory");
}

template <typename DType, typename ValType, typename PtrType>
__device__ __forceinline__ void MULTIMEM_LD(ValType& val, PtrType ptr)
{
    if constexpr (std::is_same_v<DType, half>)
    {
        asm("multimem.ld_reduce.global.add.v4.f16x2 {%0,%1,%2,%3}, [%4];"
            : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
            : "l"(ptr)
            : "memory");
    }
#ifdef ENABLE_BF16
    if constexpr (std::is_same_v<DType, __hip_bfloat16>)
    {
        asm("multimem.ld_reduce.global.add.v4.bf16x2 {%0,%1,%2,%3}, [%4];"
            : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
            : "l"(ptr)
            : "memory");
    }
#endif
}

// All MC kernels here
template <typename DType, int RANKS>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc(int const op, int const flagoffset,
    int const firstrank, int const myrank, int const gpustep, size_t const lineoffset, int const numlines,
    void** commbuff, int const handleridx, float4* mc_ptr)
{
    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = (*reduceidptr) + 1;
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;

        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
        reduce_id++;
    }
    __syncthreads();
#define UNROLL_MC 8
    int const loop_step0 = blockDim.x * gridDim.x * RANKS;
    int const loop_step = loop_step0 * UNROLL_MC;
    int const start_elem = threadIdx.x + blockDim.x * (myrank + RANKS * blockIdx.x);
    int const end_elem = max(start_elem, numlines);
    int const aligned_elem = ((end_elem - start_elem) / loop_step) * loop_step;
    int const end_aligned = start_elem + aligned_elem;

    for (int line = start_elem; line < end_aligned; line += loop_step)
    {
        uint4 val[UNROLL_MC];
#pragma unroll
        for (int i = 0; i < UNROLL_MC; i++)
            MULTIMEM_LD<DType>(val[i], mc_ptr + (lineoffset + line + i * loop_step0));
#pragma unroll
        for (int i = 0; i < UNROLL_MC; i++)
            MULTIMEM_ST(val[i], mc_ptr + (lineoffset + line + i * loop_step0));
    }
    for (int line = end_aligned; line < end_elem; line += loop_step0)
    {
        uint4 val;
        MULTIMEM_LD<DType>(val, mc_ptr + (lineoffset + line));
        MULTIMEM_ST(val, mc_ptr + (lineoffset + line));
    }
    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence_system();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // fp16 inplace reduce kernel (Hopper) MC

#else
template <typename DType, int RANKS>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc(int const op, int const flagoffset,
    int const firstrank, int const myrank, int const gpustep, size_t const lineoffset, int const numlines,
    void** commbuff, int const handleridx, float4* mc_ptr)
{
    printf("userbuffer based kernels not implemented when SM < 90\n");
    asm volatile("brkpt;\n");
}

#endif

#define callranks(x)                                                                                                   \
    if (ar_nvsize == x)                                                                                                \
    {                                                                                                                  \
        int arg1 = op - MAX_OPS,                                                                                       \
            arg2 = REG0_OFFSET(comm) - (op == userbuffers_allreduceop_nonsharp ? 2 : 1) * REG0_SINGLENODE + MAX_OPS,   \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8;                                                                                      \
        int arg7 = elements / 8;                                                                                       \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* kernelArgs[]                                                                                             \
            = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2), reinterpret_cast<void*>(&arg3),         \
                reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5), reinterpret_cast<void*>(&arg6),        \
                reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8), reinterpret_cast<void*>(&arg9)};       \
        CUDA_CHECK(hipLaunchKernelExC(&cfg,                                                                      \
            (void*) (comm->use_rr_kernel ? userbuffers_fp16_sum_inplace_gpu_rr<DType, x>                               \
                                         : userbuffers_fp16_sum_inplace_gpu_rw<DType, x>),                             \
            kernelArgs));                                                                                              \
    }

#define callranksMC(x)                                                                                                 \
    if (ar_nvsize == x)                                                                                                \
    {                                                                                                                  \
        int arg1 = op - MAX_OPS,                                                                                       \
            arg2 = REG0_OFFSET(comm) - (op == userbuffers_allreduceop_nonsharp ? 2 : 1) * REG0_SINGLENODE + MAX_OPS,   \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8;                                                                                      \
        int arg7 = elements / 8;                                                                                       \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10)};                                          \
        CUDA_CHECK(                                                                                               \
            hipLaunchKernelExC(&cfg, (void*) (userbuffers_fp16_sum_inplace_gpu_mc<DType, x>), kernelArgs));           \
    }

struct LaunchConfig
{
    LaunchConfig(communicator* comm, int sms, int threads, hipStream_t stream)
    {
        cfg.gridDim = sms;
        cfg.blockDim = threads;
        cfg.dynamicSmemBytes = 0;
        cfg.stream = stream;
        attribute[0].id = hipLaunchAttributeCooperative;
        attribute[1].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attribute[1].val.programmaticStreamSerializationAllowed = comm->pdl_launch;
        attribute[2].id = cudaLaunchAttributeClusterDimension;
        attribute[2].val.clusterDim.x = sms % comm->cga_size == 0 ? comm->cga_size : 1;
        attribute[2].val.clusterDim.y = 1;
        attribute[2].val.clusterDim.z = 1;
        cfg.attrs = attribute;
        cfg.numAttrs = comm->sm_arch >= 9 ? 3 : 1;
    }

    hipLaunchConfig_t& get()
    {
        return cfg;
    }

    hipLaunchConfig_t cfg;
    hipLaunchAttribute attribute[3];
};

template <typename DType>
__inline__ __device__ float compute_rmsnorm2(float val, float s_variance, DType const* gamma, DType const* beta, int i)
{
    float ret = val * s_variance * (float) (gamma[i]);
    if (beta != nullptr)
    {
        ret = ret + (float) (beta[i]);
    }
    return ret;
}

#define shard_tokens(ntokens, nranks, myrank)                                                                          \
    int first_token = 0, my_tokens;                                                                                    \
    {                                                                                                                  \
        int remapped_rank = myrank;                                                                                    \
        my_tokens = ntokens / nranks;                                                                                  \
        int extra_tokens = ntokens % nranks;                                                                           \
        first_token = remapped_rank * my_tokens;                                                                       \
        first_token += remapped_rank < extra_tokens ? remapped_rank : extra_tokens;                                    \
        if (remapped_rank < extra_tokens)                                                                              \
            my_tokens++;                                                                                               \
    }

#if __CUDA_ARCH__ >= 900

template <typename DType, int UNROLL_NLINES>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant(int const op,
    int const flagoffset, int const firstrank, int const myrank, int const gpustep, size_t const lineoffset,
    int const numlines, void** commbuff, int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma,
    float const eps, int const RANKS, float2* mc_ptr_out, size_t const out_lineoffset, float const* scale,
    uint4* residual_in, uint4* residual_out, int res_offset)
{
    cudaTriggerProgrammaticLaunchCompletion();
    float const sf = 1.f / (*scale);
    __shared__ float s_variance;
    int hidden_dim = blockDim.x * UNROLL_NLINES * sizeof(int4) / sizeof(DType);

    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = (*reduceidptr) + 1;
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
        cudaGridDependencySynchronize();
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
        reduce_id++;
    }
    __syncthreads();

    int const loop_step0 = blockDim.x;
    int const loop_step = loop_step0 * UNROLL_NLINES * gridDim.x;
    int const start_elem = threadIdx.x + blockDim.x * blockIdx.x * UNROLL_NLINES;
    int const end_elem = max(start_elem, numlines);

    for (int line = start_elem; line < end_elem; line += loop_step)
    {
        uint4 val[UNROLL_NLINES];
        DType* x = reinterpret_cast<DType*>(&val[0]);
#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            MULTIMEM_LD<DType>(val[i], mc_ptr + (lineoffset + line + i * loop_step0));

        if (residual_in != nullptr)
        {
#pragma unroll
            for (int i = 0; i < UNROLL_NLINES; i++)
            {
                uint4 resval = residual_in[res_offset + line + i * loop_step0];
                DType* y = reinterpret_cast<DType*>(&resval);
#pragma unroll
                for (int j = 0; j < 8; j++)
                    x[i * 8 + j] += y[j];
                residual_out[res_offset + line + i * loop_step0] = val[i];
            }
        }

        float local_var_sum = 0.0f;
        for (int j = 0; j < UNROLL_NLINES * sizeof(int4) / sizeof(DType); j++)
            local_var_sum += (float) (x[j]) * (float) (x[j]);

        float packed[1] = {local_var_sum};
        blockReduceSumV2<float, 1>(packed);
        float variance = packed[0];

        if (threadIdx.x == 0)
        {
            variance = (variance / hidden_dim); // Var[x] = E[x²]
            s_variance = rsqrtf(variance + eps);
        }
        __syncthreads();

        int i = 0;
        uint2 valout;
        __hip_fp8_e4m3_fnuz* y = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(&valout);
#pragma unroll
        for (int g = 0; g < UNROLL_NLINES; g++)
        {
#pragma unroll
            for (int j = 0; j < sizeof(int4) / sizeof(DType); j++)
            {
                y[j] = cuda_cast<__hip_fp8_e4m3_fnuz>(sf
                    * compute_rmsnorm2<DType>((float) x[i], s_variance, gamma, beta,
                        (threadIdx.x + g * loop_step0) * sizeof(int4) / sizeof(DType) + j));
                i++;
            }
            MULTIMEM_ST2(valout, mc_ptr_out + (out_lineoffset + line + g * loop_step0));
        }
    }
    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence_system();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // quant kernel fp16->fp8 twoshot

template <typename DType, int UNROLL_NLINES>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_oneshot(int const op,
    int const flagoffset, int const firstrank, int const myrank, int const gpustep, size_t const lineoffset,
    int const numlines, void** commbuff, int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma,
    float const eps, int const RANKS, uint2* mc_ptr_out, size_t const out_lineoffset, float const* scale,
    uint4* residual_in, uint4* residual_out, int res_offset)
{
    cudaTriggerProgrammaticLaunchCompletion();
    float const sf = 1.f / (*scale);
    __shared__ float s_variance;
    int hidden_dim = blockDim.x * UNROLL_NLINES * sizeof(int4) / sizeof(DType);

    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = (*reduceidptr) + 1;
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
        cudaGridDependencySynchronize();
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    __syncthreads();

    int const loop_step0 = blockDim.x;
    int const loop_step = loop_step0 * UNROLL_NLINES * gridDim.x;
    int const start_elem = threadIdx.x + blockDim.x * blockIdx.x * UNROLL_NLINES;
    int const end_elem = max(start_elem, numlines);

    for (int line = start_elem; line < end_elem; line += loop_step)
    {
        uint4 val[UNROLL_NLINES];
        DType* x = reinterpret_cast<DType*>(&val[0]);
#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            MULTIMEM_LD<DType>(val[i], mc_ptr + (lineoffset + line + i * loop_step0));

        if (residual_in != nullptr)
        {
#pragma unroll
            for (int i = 0; i < UNROLL_NLINES; i++)
            {
                uint4 resval = residual_in[res_offset + line + i * loop_step0];
                DType* y = reinterpret_cast<DType*>(&resval);
#pragma unroll
                for (int j = 0; j < 8; j++)
                    x[i * 8 + j] += y[j];
                residual_out[res_offset + line + i * loop_step0] = val[i];
            }
        }

        float local_var_sum = 0.0f;
        for (int j = 0; j < UNROLL_NLINES * sizeof(int4) / sizeof(DType); j++)
            local_var_sum += (float) (x[j]) * (float) (x[j]);

        float packed[1] = {local_var_sum};
        blockReduceSumV2<float, 1>(packed);
        float variance = packed[0];

        if (threadIdx.x == 0)
        {
            variance = (variance / hidden_dim); // Var[x] = E[x²]
            s_variance = rsqrtf(variance + eps);
        }
        __syncthreads();

        int i = 0;
        uint2 valout;
        __hip_fp8_e4m3_fnuz* y = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(&valout);

#pragma unroll
        for (int g = 0; g < UNROLL_NLINES; g++)
        {
#pragma unroll
            for (int j = 0; j < sizeof(int4) / sizeof(DType); j++)
            {
                y[j] = cuda_cast<__hip_fp8_e4m3_fnuz>(sf
                    * compute_rmsnorm2<DType>((float) x[i], s_variance, gamma, beta,
                        (threadIdx.x + g * loop_step0) * sizeof(int4) / sizeof(DType) + j));
                i++;
            }
            mc_ptr_out[out_lineoffset + line + g * loop_step0] = valout;
        }
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // quant kernel fp16->fp8 oneshot

template <typename DType, int UNROLL_NLINES>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_mc_res_allgather(int const op, int const flagoffset, int const firstrank,
        int const myrank, int const gpustep, size_t const lineoffset, int const numlines, void** commbuff,
        int const handleridx, float4* mc_ptr, int const RANKS, uint4* residual_in, int res_offset)
{
    cudaTriggerProgrammaticLaunchCompletion();
    cudaGridDependencySynchronize();
    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = (*reduceidptr) + 1;
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
    }
    __syncthreads();

    int const loop_step0 = blockDim.x;
    int const loop_step = loop_step0 * UNROLL_NLINES * gridDim.x;
    int const start_elem = threadIdx.x + blockDim.x * blockIdx.x * UNROLL_NLINES;
    int const end_elem = max(start_elem, numlines);

    for (int line = start_elem; line < end_elem; line += loop_step)
    {
        uint4 val[UNROLL_NLINES];

#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            val[i] = residual_in[res_offset + line + i * loop_step0];

#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            MULTIMEM_ST(val[i], mc_ptr + (lineoffset + line + i * loop_step0));
    }
    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence_system();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // residual allgather kernel

#else
template <typename DType, int UNROLL_NLINES>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant(int const op,
    int const flagoffset, int const firstrank, int const myrank, int const gpustep, size_t const lineoffset,
    int const numlines, void** commbuff, int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma,
    float const eps, int const RANKS, float2* mc_ptr_out, size_t const out_lineoffset, float const* scale,
    uint4* residual_in, uint4* residual_out, int res_offset)
{
    printf("userbuffer based kernels not implemented when SM < 90\n");
    asm volatile("brkpt;\n");
}

template <typename DType, int UNROLL_NLINES>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_mc_res_allgather(int const op, int const flagoffset, int const firstrank,
        int const myrank, int const gpustep, size_t const lineoffset, int const numlines, void** commbuff,
        int const handleridx, float4* mc_ptr, int const RANKS, uint4* residual_in, int res_offset)
{
    printf("userbuffer based kernels not implemented when SM < 90\n");
    asm volatile("brkpt;\n");
}

template <typename DType, int UNROLL_NLINES>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_oneshot(int const op,
    int const flagoffset, int const firstrank, int const myrank, int const gpustep, size_t const lineoffset,
    int const numlines, void** commbuff, int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma,
    float const eps, int const RANKS, uint2* ptr_out, size_t const out_lineoffset, float const* scale,
    uint4* residual_in, uint4* residual_out, int res_offset)
{
    printf("userbuffer based kernels not implemented when SM < 90\n");
    asm volatile("brkpt;\n");
}

#endif

#define callranksMC_RMSNORM_QUANT(x)                                                                                   \
    if (nlines == x)                                                                                                   \
    {                                                                                                                  \
        int arg1 = op - MAX_OPS,                                                                                       \
            arg2 = REG0_OFFSET(comm) - (op == userbuffers_allreduceop_nonsharp ? 2 : 1) * REG0_SINGLENODE + MAX_OPS,   \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8 + first_token * hidden_lines;                                                         \
        int arg7 = hidden_lines * my_tokens;                                                                           \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        DType* arg11 = (DType*) beta;                                                                                  \
        DType* arg12 = (DType*) gamma;                                                                                 \
        float arg13 = eps;                                                                                             \
        int arg14 = ar_nvsize;                                                                                         \
        void* arg15 = comm->mc_ptr[out_handler];                                                                       \
        size_t arg16 = out_offset / 8 + first_token * hidden_lines;                                                    \
        float* arg17 = scalefactor;                                                                                    \
        void* arg18 = residual_in;                                                                                     \
        void* arg19 = residual_out;                                                                                    \
        int arg20 = first_token * hidden_lines;                                                                        \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10), reinterpret_cast<void*>(&arg11),          \
            reinterpret_cast<void*>(&arg12), reinterpret_cast<void*>(&arg13), reinterpret_cast<void*>(&arg14),         \
            reinterpret_cast<void*>(&arg15), reinterpret_cast<void*>(&arg16), reinterpret_cast<void*>(&arg17),         \
            reinterpret_cast<void*>(&arg18), reinterpret_cast<void*>(&arg19), reinterpret_cast<void*>(&arg20)};        \
        CUDA_CHECK(hipLaunchKernelExC(                                                                           \
            &cfg, (void*) (userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant<DType, x>), kernelArgs));                 \
    }

#define callranksMC_RMSNORM_QUANT_ONESHOT(x)                                                                           \
    if (nlines == x)                                                                                                   \
    {                                                                                                                  \
        int arg1 = op - MAX_OPS,                                                                                       \
            arg2 = REG0_OFFSET(comm) - (op == userbuffers_allreduceop_nonsharp ? 2 : 1) * REG0_SINGLENODE + MAX_OPS,   \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8;                                                                                      \
        int arg7 = elements / 8;                                                                                       \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        DType* arg11 = (DType*) beta;                                                                                  \
        DType* arg12 = (DType*) gamma;                                                                                 \
        float arg13 = eps;                                                                                             \
        int arg14 = ar_nvsize;                                                                                         \
        void* arg15 = comm->mem_ptr[out_handler];                                                                      \
        size_t arg16 = out_offset / 8;                                                                                 \
        float* arg17 = scalefactor;                                                                                    \
        void* arg18 = residual_in;                                                                                     \
        void* arg19 = residual_out;                                                                                    \
        int arg20 = 0;                                                                                                 \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10), reinterpret_cast<void*>(&arg11),          \
            reinterpret_cast<void*>(&arg12), reinterpret_cast<void*>(&arg13), reinterpret_cast<void*>(&arg14),         \
            reinterpret_cast<void*>(&arg15), reinterpret_cast<void*>(&arg16), reinterpret_cast<void*>(&arg17),         \
            reinterpret_cast<void*>(&arg18), reinterpret_cast<void*>(&arg19), reinterpret_cast<void*>(&arg20)};        \
        CUDA_CHECK(hipLaunchKernelExC(                                                                           \
            &cfg, (void*) (userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_oneshot<DType, x>), kernelArgs));         \
    }

#define callranksMC_RES_AG(x)                                                                                          \
    if (nlines == x)                                                                                                   \
    {                                                                                                                  \
        int arg1 = op - MAX_OPS,                                                                                       \
            arg2 = REG0_OFFSET(comm) - (op == userbuffers_allreduceop_nonsharp ? 2 : 1) * REG0_SINGLENODE + MAX_OPS,   \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8 + first_token * hidden_lines;                                                         \
        int arg7 = hidden_lines * my_tokens;                                                                           \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        int arg11 = ar_nvsize;                                                                                         \
        uint4* arg12 = (uint4*) residual_in;                                                                           \
        int arg13 = first_token * hidden_lines;                                                                        \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10), reinterpret_cast<void*>(&arg11),          \
            reinterpret_cast<void*>(&arg12), reinterpret_cast<void*>(&arg13)};                                         \
        CUDA_CHECK(hipLaunchKernelExC(                                                                           \
            &cfg, (void*) (userbuffers_fp16_sum_inplace_gpu_mc_res_allgather<DType, x>), kernelArgs));                 \
    }

template <typename DType>
int allreduce2_userbuff_inplace_gpu(int const maxcredit, int const handler, size_t const offset, size_t const elements,
    int const blocksize, communicator* comm, hipStream_t stream, int op)
{
    // schedule GPU kernel only
    // CPU/SHARP part is responsibility of caller
    int const ar_firstgpu = op == userbuffers_allreduceop_nonsharp ? comm->ar_firstgpu : comm->ar2_firstgpu;
    int const ar_step = op == userbuffers_allreduceop_nonsharp2 ? 1 : comm->ar2_nvsize;
    int const ar_nvsize = op == userbuffers_allreduceop_nonsharp ? comm->ar_nvsize : comm->ar2_nvsize;
    int const ar_nvrank = op == userbuffers_allreduceop_nonsharp ? comm->ar_nvrank : comm->ar2_nvrank;

    if (elements < 8)
        return 0;
    int sms = ar_nvsize == 1 ? 2 : comm->sms;
    int warps = comm->threads / 32;
    if (warps < ar_nvsize)
        warps = ar_nvsize;
    LaunchConfig launch_config(comm, sms, warps * 32, stream);
    auto& cfg = launch_config.get();
    if (op == userbuffers_allreduceop_nonsharp2 && comm->use_mc && (comm->memflags[handler] & UB_MEM_MC_CREATED))
    {
        callranksMC(2) callranksMC(4) callranksMC(8)
#ifdef MNNVL
            callranksMC(16) callranksMC(32)
#endif
    }
    else
    {
        callranks(2) callranks(4) callranks(8)
#ifdef MNNVL
            callranks(16) callranks(32)
#endif
    }

    return sms;
}

template <typename DType>
void allreduce_nonsharp_inplace(
    int const handler, size_t const offset, size_t const elements, communicator* comm, hipStream_t stream, int op)
{
    if (elements < 64)
        return;
    int blocksize = elements * 2;
    int maxcredit = 0;
    int sms = allreduce2_userbuff_inplace_gpu<DType>(maxcredit, handler, offset, elements, blocksize, comm, stream, op);
}

template <typename DType>
void allreduce2_userbuff_inplace(
    int const handler, size_t const offset, size_t const elements, communicator* comm, hipStream_t stream)
{
    allreduce_nonsharp_inplace<DType>(handler, offset, elements, comm, stream, userbuffers_allreduceop_nonsharp2);
}

template <typename DType>
int allreduce2_userbuff_inplace_rmsnorm_quant(int const handler, size_t const offset, int const out_handler,
    size_t const out_offset, size_t const elements, int const hidden_size, void* beta, void* gamma, float eps,
    float* scalefactor, void* residual_in, void* residual_out, communicator* comm, hipStream_t stream)
{
    // schedule GPU kernel only
    // CPU/SHARP part is not supported yet;
    int op = userbuffers_allreduceop_nonsharp2;
    int const ar_firstgpu = op == userbuffers_allreduceop_nonsharp ? comm->ar_firstgpu : comm->ar2_firstgpu;
    int const ar_step = op == userbuffers_allreduceop_nonsharp2 ? 1 : comm->ar2_nvsize;
    int const ar_nvsize = op == userbuffers_allreduceop_nonsharp ? comm->ar_nvsize : comm->ar2_nvsize;
    int const ar_nvrank = op == userbuffers_allreduceop_nonsharp ? comm->ar_nvrank : comm->ar2_nvrank;

    if (elements % hidden_size)
        return 0;
    CHECK(hidden_size % 8 == 0);
    int hidden_lines = hidden_size / 8;
    shard_tokens(elements / hidden_size, ar_nvsize, ar_nvrank);

    int sms = ar_nvsize == 1 ? 2 : comm->sms;
    int nthreads = hidden_size / 8;
    int nlines = 1;
    while (nthreads > 1024)
    {
        nlines++;
        CHECK(nlines <= 4);
        if ((hidden_size / 8) % nlines == 0)
            nthreads = ((hidden_size / 8)) / nlines;
    }

    LaunchConfig launch_config(comm, sms, nthreads, stream);
    auto& cfg = launch_config.get();
    if (op == userbuffers_allreduceop_nonsharp2 && comm->use_mc && (comm->memflags[handler] & UB_MEM_MC_CREATED))
    {
        if (comm->oneshot != 0 && (elements * ar_nvsize <= 131072))
        {
            callranksMC_RMSNORM_QUANT_ONESHOT(1) callranksMC_RMSNORM_QUANT_ONESHOT(2)
                callranksMC_RMSNORM_QUANT_ONESHOT(3) callranksMC_RMSNORM_QUANT_ONESHOT(4)
        }
        else
        {
            callranksMC_RMSNORM_QUANT(1) callranksMC_RMSNORM_QUANT(2) callranksMC_RMSNORM_QUANT(3)
                callranksMC_RMSNORM_QUANT(4)
        }
    }
    else
    {
        CHECK(0);
    }

    return sms;
}

template <typename DType>
int allgather2_userbuff_residual(int const handler, size_t const offset, size_t const elements, int const hidden_size,
    void* residual_in, communicator* comm, hipStream_t stream)
{
    // schedule GPU kernel only
    // CPU/SHARP part is not supported yet;
    if (comm->oneshot != 0 && (elements * comm->ar2_nvsize <= 131072))
    {
        CUDA_CHECK(hipMemcpyAsync(reinterpret_cast<uint8_t*>(comm->mem_ptr[handler]) + (offset * 2), residual_in,
            elements * 2, hipMemcpyDeviceToDevice, stream));
        return 0;
    }
    int op = userbuffers_allreduceop_nonsharp2;
    int const ar_firstgpu = op == userbuffers_allreduceop_nonsharp ? comm->ar_firstgpu : comm->ar2_firstgpu;
    int const ar_step = op == userbuffers_allreduceop_nonsharp2 ? 1 : comm->ar2_nvsize;
    int const ar_nvsize = op == userbuffers_allreduceop_nonsharp ? comm->ar_nvsize : comm->ar2_nvsize;
    int const ar_nvrank = op == userbuffers_allreduceop_nonsharp ? comm->ar_nvrank : comm->ar2_nvrank;

    if (elements % hidden_size)
        return 0;
    CHECK(hidden_size % 8 == 0);
    int hidden_lines = hidden_size / 8;
    shard_tokens(elements / hidden_size, ar_nvsize, ar_nvrank);

    int sms = ar_nvsize == 1 ? 2 : comm->sms;
    int nthreads = hidden_size / 8;
    int nlines = 1;
    while (nthreads > 1024)
    {
        nlines++;
        CHECK(nlines <= 4);
        if ((hidden_size / 8) % nlines == 0)
            nthreads = ((hidden_size / 8)) / nlines;
    }
    LaunchConfig launch_config(comm, sms, nthreads, stream);
    auto& cfg = launch_config.get();
    if (op == userbuffers_allreduceop_nonsharp2 && comm->use_mc && (comm->memflags[handler] & UB_MEM_MC_CREATED))
    {
        callranksMC_RES_AG(1) callranksMC_RES_AG(2) callranksMC_RES_AG(3) callranksMC_RES_AG(4)
    }
    else
    {
        CHECK(0);
    }

    return sms;
}

void allreduce2_userbuff_inplace_impl(int const handler, size_t const offset, size_t const elements,
    nvinfer1::DataType dataType, communicator* comm, hipStream_t stream)
{
    switch (dataType)
    {
    case nvinfer1::DataType::kHALF: allreduce2_userbuff_inplace<half>(handler, offset, elements, comm, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        allreduce2_userbuff_inplace<__hip_bfloat16>(handler, offset, elements, comm, stream);
        break;
#endif
    default: THROW("Unsupported dataType for allreduce2_userbuff_inplace_impl");
    }
}

int allgather2_userbuff_residual_impl(int const handler, size_t const offset, size_t const elements,
    int const hidden_size, void* residual, nvinfer1::DataType dataType, communicator* comm, hipStream_t stream)
{
    switch (dataType)
    {
    case nvinfer1::DataType::kHALF:
        return allgather2_userbuff_residual<half>(handler, offset, elements, hidden_size, residual, comm, stream);
        break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        return allgather2_userbuff_residual<__hip_bfloat16>(
            handler, offset, elements, hidden_size, residual, comm, stream);
        break;
#endif
    default: THROW("Unsupported dataType for allgather2_userbuff_residual_impl");
    }
}

int allreduce2_userbuff_inplace_rmsnorm_quant_impl(int const handler, size_t const offset, int const out_handler,
    size_t const out_offset, size_t const elements, int const hidden_size, void* beta, void* gamma, float eps,
    float* scalefactor, void* residual_in, void* residual_out, nvinfer1::DataType dataType, communicator* comm,
    hipStream_t stream)
{
    switch (dataType)
    {
    case nvinfer1::DataType::kHALF:
        return allreduce2_userbuff_inplace_rmsnorm_quant<half>(handler, offset, out_handler, out_offset, elements,
            hidden_size, beta, gamma, eps, scalefactor, residual_in, residual_out, comm, stream);
        break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        return allreduce2_userbuff_inplace_rmsnorm_quant<__hip_bfloat16>(handler, offset, out_handler, out_offset,
            elements, hidden_size, beta, gamma, eps, scalefactor, residual_in, residual_out, comm, stream);
        break;
#endif
    default: THROW("Unsupported dataType for allreduce2_userbuff_inplace_rmsnorm_quant_impl");
    }
}
} // namespace sugesstify::kernels::ub
