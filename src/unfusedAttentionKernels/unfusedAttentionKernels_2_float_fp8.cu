#include "hip/hip_runtime.h"


#include "unfusedAttentionKernels_2_template.h"

namespace suggestify
{
namespace kernels
{

INSTANTIATE_ATTENTION_INPUT_OUTPUT_PROCESSING(float, __hip_fp8_e4m3_fnuz, KVBlockArray);
INSTANTIATE_ATTENTION_INPUT_OUTPUT_PROCESSING(float, __hip_fp8_e4m3_fnuz, KVLinearBuffer);

} // namespace kernels
} // namespace suggestify
