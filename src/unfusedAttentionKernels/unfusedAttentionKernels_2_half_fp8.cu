#include "hip/hip_runtime.h"


#include "unfusedAttentionKernels_2_template.h"

namespace suggestify
{
namespace kernels
{

INSTANTIATE_ATTENTION_INPUT_OUTPUT_PROCESSING(half, __hip_fp8_e4m3_fnuz, KVBlockArray);
INSTANTIATE_ATTENTION_INPUT_OUTPUT_PROCESSING(half, __hip_fp8_e4m3_fnuz, KVLinearBuffer);

} // namespace kernels
} // namespace suggestify
