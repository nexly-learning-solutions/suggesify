#include "hip/hip_runtime.h"
#include "cudautil.h"
#include "data.h"
#include "exactGpu.h"
#include "mathUtil.h"
#include "output.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <ranges>
#include <span>
#include <stdexcept>
#include <string>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

namespace astdl
{
namespace knn
{

class GpuContext {
public:
    GpuContext(int device, int collectionRows, int collectionCols, int maxK, DataType dataType)
        : device(device), 
          collectionRows(collectionRows),
          collectionCols(collectionCols),
          maxK(maxK),
          dataType(dataType)
    {
        CHECK_ERR(hipSetDevice(device));
        CHECK_ERR(hipMallocManaged(&dInputBatchBuffer.data, dInputBatchBuffer.getSizeInBytes()));
        CHECK_ERR(hipMallocManaged(&dProducts.data, dProducts.getSizeInBytes()));
        CHECK_ERR(hipMallocManaged(&dResultScores.data, dResultScores.getSizeInBytes()));
        CHECK_ERR(hipMallocManaged(&dResultIndexes.data, dResultIndexes.getSizeInBytes()));
        CHECK_ERR(hipMallocPitch(&dCollectionPartition.data, &dCollectionPartition.pitch, 
                             dCollectionPartition.getSizeInBytes(), dCollectionPartition.numRows));

        CHECK_ERR(hipStreamCreate(&stream));
        CHECK_ERR(hipblasCreate(&handle));
        CHECK_ERR(hipEventCreate(&startEvent));
        CHECK_ERR(hipEventCreate(&stopEvent));

        CHECK_ERR(hipMallocManaged(&dHeapScores, heapSize * sizeof(float)));
        CHECK_ERR(hipMallocManaged(&dHeapIndexes, heapSize * sizeof(uint32_t)));

        elapsedTopK = 0.0f;
        elapsedSgemm = 0.0f;
    }

    ~GpuContext()
    {
        CHECK_ERR(hipFree(dInputBatchBuffer.data));
        CHECK_ERR(hipFree(dProducts.data));
        CHECK_ERR(hipFree(dResultScores.data));
        CHECK_ERR(hipFree(dResultIndexes.data));
        CHECK_ERR(hipFree(dCollectionPartition.data));
        CHECK_ERR(hipStreamDestroy(stream));
        CHECK_ERR(hipblasDestroy(handle));
        CHECK_ERR(hipEventDestroy(startEvent));
        CHECK_ERR(hipEventDestroy(stopEvent));
        CHECK_ERR(hipFree(dHeapScores));
        CHECK_ERR(hipFree(dHeapIndexes));
    }

    void performMatrixMultiplication(int batchSize)
    {
        static constexpr hipblasOperation_t transa = HIPBLAS_OP_N;
        static constexpr hipblasOperation_t transb = HIPBLAS_OP_N;
        static constexpr float alpha = 1.0f;
        static constexpr float beta = 0.0f;

        hipDataType aType, bType, cType = HIP_R_32F;
        if (dataType == DataType::FP16)
        {
            aType = HIP_R_16F;
            bType = HIP_R_16F;
            CHECK_ERR(cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dCollectionPartition.numRows, 
                              batchSize, dCollectionPartition.numColumns, &alpha, 
                              dCollectionPartition.data, aType, dCollectionPartition.numRows,
                              dInputBatchBuffer.data, bType, dInputBatchBuffer.numColumns, &beta,
                              dProducts.data, cType, dProducts.numRows,
                              HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP, 
                              hipStream_t(stream))); 
        }
        else if (dataType == DataType::FP32)
        {
            aType = HIP_R_32F;
            bType = HIP_R_32F;
            CHECK_ERR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dCollectionPartition.numRows, 
                              batchSize, dCollectionPartition.numColumns, &alpha, 
                              dCollectionPartition.data, dCollectionPartition.numRows,
                              dInputBatchBuffer.data, dInputBatchBuffer.numColumns, &beta,
                              dProducts.data, dProducts.numRows, 
                              hipStream_t(stream)));
        }

        CHECK_ERR(hipEventRecord(startEvent, stream));
        CHECK_ERR(hipEventRecord(stopEvent, stream));
        CHECK_ERR(hipStreamSynchronize(stream));

        float elapsedTime;
        CHECK_ERR(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));
        elapsedSgemm = elapsedTime;
    }

    void performTopK(int batchSize)
    {
        const int blockDim = 256;
        const int numBlocks = (batchSize + blockDim - 1) / blockDim;

        CHECK_ERR(hipEventRecord(startEvent, stream));
        topKShared<<<numBlocks, blockDim>>>(static_cast<float*>(dProducts.data), 
                                    dResultScores.data, dResultIndexes.data, 
                                    batchSize, dProducts.numColumns, 
                                    collectionRows, maxK);
        CHECK_ERR(hipDeviceSynchronize());
        CHECK_ERR(hipEventRecord(stopEvent, stream));
        CHECK_ERR(hipStreamSynchronize(stream));

        float elapsedTime;
        CHECK_ERR(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));
        elapsedTopK = elapsedTime;
    }

    template <typename T>
    void copyInputToDevice(const T* inputs, size_t inputSize)
    {
        if (dataType == DataType::FP16)
        {
            astdl::math::kFloatToHalf(inputs, inputSize * sizeof(T),
                static_cast<half*>(dInputBatchBuffer.data),
                static_cast<float*>(dInputBatchBuffer.data),
                dInputBatchBuffer.getSizeInBytes());
        }
        else if (dataType == DataType::FP32)
        {
        }
        else
        {
            throw std::runtime_error("Unknown data type");
        }
    }

    void copyResultsToHost(float* hResultScores, uint32_t* hResultIndexes, int batchSize)
    {
        CHECK_ERR(hipMemcpy(hResultScores, dResultScores.data, batchSize * maxK * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_ERR(hipMemcpy(hResultIndexes, dResultIndexes.data, batchSize * maxK * sizeof(uint32_t), hipMemcpyDeviceToHost));
    }

    float getElapsedSgemm() const { return elapsedSgemm; }
    float getElapsedTopK() const { return elapsedTopK; }

    int device; 
    int collectionRows;
    int collectionCols;
    int maxK;
    DataType dataType;

    hipblasHandle_t handle;
    Matrix dCollectionPartition; 
    Matrix dInputBatchBuffer;
    Matrix dProducts;
    Matrix dResultScores;
    Matrix dResultIndexes;
    hipStream_t stream; 
    hipEvent_t startEvent, stopEvent;

    float* dHeapScores;
    uint32_t* dHeapIndexes;
    size_t heapSize = 256;

    float elapsedTopK;
    float elapsedSgemm;
};

class KnnExactGpu {
public:
    KnnExactGpu(KnnData* data) : data(data)
    {
        gpuContexts.resize(data->numGpus);
        for (int device = 0; device < data->numGpus; ++device)
        {
            gpuContexts[device] = std::make_unique<GpuContext>(
                device, data->dCollectionPartition.numRows / data->numGpus + (device < data->dCollectionPartition.numRows % data->numGpus ? 1 : 0),
                data->dCollectionPartition.numColumns, data->maxK, data->dataType);

            CHECK_ERR(hipMemcpy(gpuContexts[device]->dCollectionPartition.data, data->hCollectionPartitions[0].data + device * (data->dCollectionPartition.numRows / data->numGpus) * data->dCollectionPartition.numColumns, 
                                 gpuContexts[device]->dCollectionPartition.getSizeInBytes(), hipMemcpyHostToDevice));
        }

        if (usePinnedMemory)
        {
            CHECK_ERR(hipHostAlloc(&pinnedInputData, data->dCollectionPartition.numColumns * data->batchSize * sizeof(float), hipHostMallocDefault));
        }
    }

    ~KnnExactGpu() 
    {
        if (usePinnedMemory)
        {
            CHECK_ERR(hipHostFree(pinnedInputData));
        }
    }

    void search(int k, float const* inputs, int size, std::string* keys, float* scores)
    {
        if (k > data->maxK)
        {
            throw std::invalid_argument("k = " + std::to_string(k) + " is > maxK = " + std::to_string(data->maxK));
        }

        elapsedTopK.resize(data->numGpus);
        elapsedSgemm.resize(data->numGpus);

        int batchSize = data->batchSize; 

        for (int device = 0; device < data->numGpus; ++device)
        {
            gpuContexts[device]->copyInputToDevice(inputs, batchSize * data->dCollectionPartition.numColumns);
            gpuContexts[device]->performMatrixMultiplication(batchSize);
            gpuContexts[device]->performTopK(batchSize);

            elapsedSgemm[device] = gpuContexts[device]->getElapsedSgemm();
            elapsedTopK[device] = gpuContexts[device]->getElapsedTopK();
        }

        float avgSgemmTime = 0.0f;
        for (int device = 0; device < data->numGpus; ++device)
        {
            avgSgemmTime += elapsedSgemm[device];
        }
        avgSgemmTime /= data->numGpus; 

        if (avgSgemmTime > 0.01f)
        {
            batchSize = std::max(1, batchSize / 2);
        } 

        for (int i = 0; i < size; i += batchSize)
        {
            int currentBatchSize = std::min(batchSize, size - i);

            if (usePinnedMemory)
            {
                CHECK_ERR(hipMemcpy(pinnedInputData, inputs + i * data->dCollectionPartition.numColumns,
                                     currentBatchSize * data->dCollectionPartition.numColumns * sizeof(float), hipMemcpyHostToDevice));
            }

            for (int device = 0; device < data->numGpus; ++device)
            {
                if (usePinnedMemory)
                {
                    CHECK_ERR(hipMemcpyAsync(gpuContexts[device]->dInputBatchBuffer.data, pinnedInputData,
                                         currentBatchSize * data->dCollectionPartition.numColumns * sizeof(float), hipMemcpyDeviceToDevice, gpuContexts[device]->stream));
                }
                else
                {
                    gpuContexts[device]->copyInputToDevice(inputs + i * data->dCollectionPartition.numColumns,
                                                     currentBatchSize * data->dCollectionPartition.numColumns);
                }

                gpuContexts[device]->performMatrixMultiplication(currentBatchSize);
                gpuContexts[device]->performTopK(currentBatchSize);

                gpuContexts[device]->copyResultsToHost(data->hResultScores.data + i * k, data->hResultIndexes.data + i * k, currentBatchSize);
            }

            for (int device = 0; device < data->numGpus; ++device)
            {
                CHECK_ERR(hipStreamSynchronize(gpuContexts[device]->stream));
            }

            mergeResults(data, currentBatchSize, k);

            CHECK_ERR(hipMemcpy(scores + i * k, data->hResultScores.data, currentBatchSize * k * sizeof(float), hipMemcpyHostToDevice));
        }

        for (int device = 0; device < data->numGpus; ++device)
        {
            std::cout << "GPU " << device << ": "
                      << "Sgemm Time: " << elapsedSgemm[device] << "ms"
                      << " Top-K Time: " << elapsedTopK[device] << "ms" << std::endl;
        }
    }

private:
    void mergeResults(KnnData* data, int batchSize, int maxK)
    {
        const size_t blockDim = 256;
        const size_t numBlocks = (data->numGpus + blockDim - 1) / blockDim;

        thrust::device_vector<float> dSharedScores(data->numGpus * maxK * batchSize);
        thrust::device_vector<uint32_t> dSharedIndexes(data->numGpus * maxK * batchSize);

        #pragma omp parallel for
        for (int device = 0; device < data->numGpus; ++device)
        {
            CHECK_ERR(hipMemcpyAsync(thrust::raw_pointer_cast(dSharedScores.data()) + device * maxK * batchSize, 
                                   gpuContexts[device]->dResultScores.data, 
                                   maxK * batchSize * sizeof(float), hipMemcpyDeviceToDevice, gpuContexts[device]->stream));
            CHECK_ERR(hipMemcpyAsync(thrust::raw_pointer_cast(dSharedIndexes.data()) + device * maxK * batchSize, 
                                   gpuContexts[device]->dResultIndexes.data, 
                                   maxK * batchSize * sizeof(uint32_t), hipMemcpyDeviceToDevice, gpuContexts[device]->stream));
            CHECK_ERR(hipStreamSynchronize(gpuContexts[device]->stream));
        }

        for (int i = 0; i < batchSize; ++i)
        {
            thrust::device_ptr<float> dScoresPtr = thrust::raw_pointer_cast(dSharedScores.data()) + i * data->numGpus * maxK;
            thrust::device_ptr<uint32_t> dIndexesPtr = thrust::raw_pointer_cast(dSharedIndexes.data()) + i * data->numGpus * maxK;
            thrust::sort_by_key(dScoresPtr, dScoresPtr + data->numGpus * maxK, dIndexesPtr);
            thrust::copy(dScoresPtr, dScoresPtr + maxK, data->hResultScores.data + i * maxK);
            thrust::copy(dIndexesPtr, dIndexesPtr + maxK, data->hResultIndexes.data + i * maxK);
        }
    }

    KnnData* data;

    std::vector<std::unique_ptr<GpuContext>> gpuContexts;

    std::vector<float> elapsedTopK;
    std::vector<float> elapsedSgemm;


    float* pinnedInputData = nullptr; 
    bool usePinnedMemory = true;
};
}
}