#include "hip/hip_runtime.h"


#include "../common/assert.h"
#include "../common/cudaTypeUtils.cuh"
#include "../common/cudaUtils.h"
#include "../common/quantTypeUtils.cuh"
#include "../common/reduceKernelUtils.cuh"
#include "quantization.h"
#include <float.h>

using namespace suggestify::common;

namespace suggestify
{
namespace kernels
{

__global__ void quantizedKernel(char4* dst, float4 const* src, int64_t const sizeDiv4, float const* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        float const scale = __ldg(scalePtr);
        char4 tmp;
        float4 const floatTmp = __ldg(src + idx);
        tmp.x = cuda_cast<int8_t>(floatTmp.x * scale);
        tmp.y = cuda_cast<int8_t>(floatTmp.y * scale);
        tmp.z = cuda_cast<int8_t>(floatTmp.z * scale);
        tmp.w = cuda_cast<int8_t>(floatTmp.w * scale);
        dst[idx] = tmp;
    }
}

__global__ void quantizedKernel(char4* dst, half2 const* src, int64_t const sizeDiv4, float const* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        float const scale = __ldg(scalePtr);
        char4 tmp;
        int srcId = idx << 1;

        uint2 const h2 = __ldg(reinterpret_cast<uint2 const*>(src + srcId));

        half2 const half2Tmp = reinterpret_cast<half2 const&>(h2.x);
        half2 const half2Tmp2 = reinterpret_cast<half2 const&>(h2.y);

        tmp.x = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.x) * scale);
        tmp.y = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.y) * scale);
        tmp.z = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.x) * scale);
        tmp.w = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.y) * scale);
        dst[idx] = tmp;
    }
}

#ifdef ENABLE_BF16
__global__ void quantizedKernel(char4* dst, __hip_bfloat162 const* src, int64_t const sizeDiv4, float const* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        float const scale = __ldg(scalePtr);
        char4 tmp;
        int srcId = idx << 1;

        uint2 const h2 = __ldg(reinterpret_cast<uint2 const*>(src + srcId));

        __hip_bfloat162 const bfloat162Tmp = reinterpret_cast<__hip_bfloat162 const&>(h2.x);
        __hip_bfloat162 const bfloat162Tmp2 = reinterpret_cast<__hip_bfloat162 const&>(h2.y);

        tmp.x = cuda_cast<int8_t>(cuda_cast<float>(bfloat162Tmp.x) * scale);
        tmp.y = cuda_cast<int8_t>(cuda_cast<float>(bfloat162Tmp.y) * scale);
        tmp.z = cuda_cast<int8_t>(cuda_cast<float>(bfloat162Tmp2.x) * scale);
        tmp.w = cuda_cast<int8_t>(cuda_cast<float>(bfloat162Tmp2.y) * scale);

        dst[idx] = tmp;
    }
}
#endif

template <typename T>
void invokeQuantization(
    int8_t* dst, T const* src, int64_t const size, float const* scalePtr, hipStream_t stream, int maxGridSize)
{
    CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    dim3 grid(std::min(numBlocks, maxGridSize));
    CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (float4 const*) src, size / 4, scalePtr);
    }
    else if (std::is_same_v<T, half>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (half2 const*) src, size / 4, scalePtr);
    }
#ifdef ENABLE_BF16
    else if (std::is_same_v<T, __hip_bfloat16>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (__hip_bfloat162 const*) src, size / 4, scalePtr);
    }
#endif
}

template void invokeQuantization<float>(
    int8_t* dst, float const* src, int64_t const size, float const* scalePtr, hipStream_t stream, int maxGridSize);

template void invokeQuantization<half>(
    int8_t* dst, half const* src, int64_t const size, float const* scalePtr, hipStream_t stream, int maxGridSize);

#ifdef ENABLE_BF16
template void invokeQuantization<__hip_bfloat16>(int8_t* dst, __hip_bfloat16 const* src, int64_t const size,
    float const* scalePtr, hipStream_t stream, int maxGridSize);
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, int NUM_ELTS>
struct DstVec
{
    static_assert("not implemented.");
};

template <>
struct DstVec<float2, 2>
{
    using Type = uint32_t;
};

template <>
struct DstVec<half2, 4>
{
    using Type = uint2;
};

#ifdef ENABLE_BF16

template <>
struct DstVec<__hip_bfloat162, 4>
{
    using Type = uint2;
};

#endif // ENABLE_BF16

template <typename T>
struct DstVec<T, 4>
{
    static_assert(sizeof(T) == 4, "not implemented.");
    using Type = uint32_t;
};

template <typename T>
struct DstVec<T, 8>
{
    static_assert(sizeof(T) == 2, "not implemented.");
    using Type = uint2;
};

////////////////////////////////////////////////////////////////////////////////////////////////////

// Helper function of getting the absMax of all elements in the vector after clamping.
// Pack two elements in order to use possible hmax2 instructions.
template <typename T>
inline __device__ void clampAndAbsMax(T& localMax, uint4& vec, T const clampMin, T const clampMax)
{
    static constexpr int NUM_ELTS = sizeof(uint4) / sizeof(T);

#pragma unroll
    for (int i = 0; i < NUM_ELTS; ++i)
    {
        T& val = reinterpret_cast<T*>(&vec)[i];
        val = cuda_clamp(val, clampMin, clampMax);
        localMax = cuda_max(localMax, cuda_abs(val));
    }
}

// Helper function of quantizing the vector and storing it to global memory.
// Pack two elements in order to use fast convert instructions.
template <typename T, typename QuantT, bool USE_SMEM>
inline __device__ void quantizeAndStore(
    QuantT* dstPtr, uint4 vec, T const clampMin, T const clampMax, float const scaleOrigQuant)
{
    static constexpr int NUM_ELTS = sizeof(uint4) / sizeof(T);

    using DstVecType = typename DstVec<T, NUM_ELTS>::Type;
    DstVecType dstVec;
#pragma unroll
    for (int i = 0; i < NUM_ELTS; ++i)
    {
        T val = reinterpret_cast<T*>(&vec)[i];
        // Values loaded from smem has already been clamped.
        if constexpr (!USE_SMEM)
        {
            val = cuda_clamp(val, clampMin, clampMax);
        }
        float2 val2 = cuda_cast<float2>(val);
        val2.x *= scaleOrigQuant;
        val2.y *= scaleOrigQuant;
        QuantT quantVal = cuda_cast<QuantT>(val2);
        reinterpret_cast<QuantT*>(&dstVec)[i] = quantVal;
    }
    // Store to destination buffer.
    *reinterpret_cast<DstVecType*>(dstPtr) = dstVec;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, typename QuantT, bool USE_SMEM>
__global__ void perTokenQuantization(QuantT* dst, T const* src, int64_t const numRows, int64_t const numCols,
    float const* clampPtr, float* scalePtr, float* sumPtr, bool hasFp8MinScaling)
{
    // Smem buffer.
    extern __shared__ uint4 smemBuffer[];

    // The clamping minimum / maximum values.
    T const clampMin = cuda_cast<T>(clampPtr ? clampPtr[0] : -FLT_MAX);
    T const clampMax = cuda_cast<T>(clampPtr ? clampPtr[1] : FLT_MAX);

    // Pack two elements in order to use higher through instructions.
    using T2 = typename packed_as<T, 2>::type;
    using QuantT2 = typename packed_as<QuantT, 2>::type;
    T2 const clampMin2 = cuda_cast<T2, T>(clampMin);
    T2 const clampMax2 = cuda_cast<T2, T>(clampMax);

    // The quantized data type's maximum value (upper-bound).
    static constexpr float MAX_QUANT_VAL = QuantTypeStaticVals<QuantT>::MAX_VAL;
    // The minimum scaling factor (lower-bound).
    static constexpr float MIN_SCALING_FACTOR = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR;
    static constexpr float MIN_SCALING_FACTOR_RCP = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR_RCP;

    // The number of elements in the packed uint4 vec.
    static constexpr int NUM_ELTS_PER_VEC = sizeof(uint4) / sizeof(T);
    static constexpr int NUM_ELTS2_PER_VEC = sizeof(uint4) / sizeof(T2);

    // The number of vectors in the column.
    int const numColVecs = numCols / NUM_ELTS_PER_VEC;
    // The vector pointers for src.
    uint4 const* srcVec = reinterpret_cast<uint4 const*>(src) + blockIdx.x * numColVecs;
    // The pointer for dst.
    QuantT* dstRow = dst + blockIdx.x * numCols;
    // T const* srcRow = src + blockIdx.x * numCols;

    T2 localMax2 = cuda_cast<T2, T>(T(1e-6f));
    float2 localSum2 = {0.f, 0.f};

    for (int i = threadIdx.x; i < numColVecs; i += blockDim.x)
    {
        uint4 vec = srcVec[i];

#pragma unroll
        for (int j = 0; j < NUM_ELTS2_PER_VEC; ++j)
        {
            T2& val2 = reinterpret_cast<T2*>(&vec)[j];
            val2 = cuda_clamp(val2, clampMin2, clampMax2);
            localMax2 = cuda_max(localMax2, cuda_abs(val2));
            // TODO: template the version that requires sum to avoid dynamic branching.
            if (sumPtr != nullptr)
            {
                localSum2.x += cuda_cast<float>(val2.x);
                localSum2.y += cuda_cast<float>(val2.y);
            }
        }
        // Avoid reloading from global memory.
        if constexpr (USE_SMEM)
        {
            smemBuffer[i] = vec;
        }
    }
    float const rowMax = blockAllReduceMax(cuda_cast<float>(cuda_max<T, T2>(localMax2)));
    if (threadIdx.x == 0)
    {
        scalePtr[blockIdx.x]
            = hasFp8MinScaling ? cuda_max(rowMax / MAX_QUANT_VAL, MIN_SCALING_FACTOR) : (rowMax / MAX_QUANT_VAL);
    }

    if (sumPtr != nullptr)
    {
        float rowSum[1] = {cuda_sum<float>(localSum2)};
        blockReduceSumV2<float, 1>(rowSum);
        if (threadIdx.x == 0)
        {
            sumPtr[blockIdx.x] = rowSum[0];
        }
    }

    float const scaleOrigQuant
        = hasFp8MinScaling ? fminf(MAX_QUANT_VAL / rowMax, MIN_SCALING_FACTOR_RCP) : MAX_QUANT_VAL / rowMax;
    for (int i = threadIdx.x; i < numColVecs; i += blockDim.x)
    {
        uint4 vec = USE_SMEM ? smemBuffer[i] : srcVec[i];
        QuantT2* dstPtr = reinterpret_cast<QuantT2*>(dstRow + i * NUM_ELTS_PER_VEC);
        quantizeAndStore<T2, QuantT2, USE_SMEM>(dstPtr, vec, clampMin2, clampMax2, scaleOrigQuant);
    }
}

// Do per-token (row) quantization from fp16/bf16/fp32 to int8/fp8_e4m3.
template <typename T, typename QuantT>
void invokePerTokenQuantization(QuantT* dst, T const* src, int64_t const numRows, int64_t const numCols,
    float const* clampPtr, float* scalePtr, float* sumPtr, QuantMode quantMode, hipStream_t stream)
{
    // each block is responsible for a single row
    dim3 const block(512);
    dim3 const grid(numRows);

    // The number of elements in the packed uint4 vec.
    static constexpr int NUM_ELTS_PER_VEC = sizeof(uint4) / sizeof(T);
    CHECK_WITH_INFO(numCols % NUM_ELTS_PER_VEC == 0, "Not supported.");

    // Cache vectors to smem to avoid reloading.
    size_t const dynamicSmemSz = numCols * sizeof(T);
    // Need to check if smem capacity is enough.
    bool useSmem = true;
    if (dynamicSmemSz >= 48 * 1024)
    {
        hipError_t res = hipFuncSetAttribute(reinterpret_cast<const void*>(
            perTokenQuantization<T), QuantT, true>, hipFuncAttributeMaxDynamicSharedMemorySize, dynamicSmemSz);
        // Fall back to reloading-reversion if smem is not enough.
        useSmem = (res == hipSuccess);
    }

    // Enable min_scaling_factor if it is fp8 rowwise per-token quantization.
    bool hasFp8MinScaling = quantMode.hasFp8RowWise();
    // Do we use smem ?
    if (useSmem)
    {
        perTokenQuantization<T, QuantT, true><<<grid, block, dynamicSmemSz, stream>>>(
            dst, src, numRows, numCols, clampPtr, scalePtr, sumPtr, hasFp8MinScaling);
    }
    else
    {
        perTokenQuantization<T, QuantT, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, clampPtr, scalePtr, sumPtr, hasFp8MinScaling);
    }
}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T, QuantT)                                                           \
    template void invokePerTokenQuantization(QuantT* dst, const T* src, const int64_t numRows, const int64_t numCols,  \
        float const* clampPtr, float* scalePtr, float* sumPtr, QuantMode quantMode, hipStream_t stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float, int8_t);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half, int8_t);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16, int8_t);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

} // namespace kernels
} // namespace suggestify
